#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <spmv_cuda.h>
#include <math.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>

#include "cuda_timer.h"

#define test_iter 2000

#define threadsPerBlock 256
#define sizeSharedMemory 1024
#define BlockDim 1024

template <unsigned int threads_per_row>
__device__ __forceinline__ float warpReduceSum(float sum)
{
    if (threads_per_row >= 32)
        sum += __shfl_down_sync(0xffffffff, sum, 16); // 0-16, 1-17, 2-18, etc.
    if (threads_per_row >= 16)
        sum += __shfl_down_sync(0xffffffff, sum, 8); // 0-8, 1-9, 2-10, etc.
    if (threads_per_row >= 8)
        sum += __shfl_down_sync(0xffffffff, sum, 4); // 0-4, 1-5, 2-6, etc.
    if (threads_per_row >= 4)
        sum += __shfl_down_sync(0xffffffff, sum, 2); // 0-2, 1-3, 4-6, 5-7, etc.
    if (threads_per_row >= 2)
        sum += __shfl_down_sync(0xffffffff, sum, 1); // 0-1, 2-3, 4-5, etc.
    return sum;
}

template <typename T>
__global__ void spmv_pcsr_kernel1(T *d_val, T *d_vector, perfSpB_index *d_cols, perfSpB_index d_nnz, T *d_middle)
{
    perfSpB_index tid = blockIdx.x * blockDim.x + threadIdx.x;
    perfSpB_index icr = blockDim.x * gridDim.x;
    while (tid < d_nnz)
    {
        d_middle[tid] = d_val[tid] * d_vector[d_cols[tid]];
        tid += icr;
    }
}
template <typename T, unsigned int ROWS_PER_BLOCK, unsigned int THREADS_PER_ROW>
__global__ void spmv_pcsr_kernel2(T *d_middle, perfSpB_index *d_ptr, perfSpB_index nrows, T *d_out)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    int b_0_row_id = (blockIdx.x * blockDim.x) / THREADS_PER_ROW;
    int thread_lane = threadIdx.x & (THREADS_PER_ROW - 1);
    perfSpB_index g_rowid = gid / THREADS_PER_ROW;

    __shared__ volatile T middle_s[sizeSharedMemory];
    int b_last_row_id = (b_0_row_id + ROWS_PER_BLOCK) > (nrows) ? (nrows) : (b_0_row_id + ROWS_PER_BLOCK);

    perfSpB_index temp = (d_ptr[b_last_row_id] - d_ptr[b_0_row_id]) / threadsPerBlock + 1;
    perfSpB_index nlen = min(perfSpB_index(temp * threadsPerBlock), perfSpB_index(sizeSharedMemory));
    T sum = 0, sum_tmp = 0;
    perfSpB_index maxlen = d_ptr[b_last_row_id];
    if (blockIdx.x == gridDim.x) 
        maxlen = d_ptr[nrows + 1];

    // 使用循环分多次加载 d_middle 到共享内存 middle_s[],每次加载量为 nlen
    if (g_rowid < nrows)
    {
        for (perfSpB_index i = d_ptr[b_0_row_id]; i < maxlen; i += nlen)
        {
            perfSpB_index index = i + tid;
            __syncthreads();
            // Load d_middle into the shared memory middle_s, 填满一个1024需要几次迭代 256×4=1024
            for (perfSpB_index j = 0; j < nlen / threadsPerBlock; j++)
            {
                if (index < maxlen)
                {
                    middle_s[tid + j * threadsPerBlock] = d_middle[index];
                    index += threadsPerBlock;
                }
            }
            __syncthreads();
            // 填满了一个1024，计算这个1024
            //  Sum up the elements for a row
            //TODO: ↓  最后一块 出  bug 
            sum_tmp = 0;
            if (d_ptr[g_rowid + 1] > i && d_ptr[g_rowid] <= i + nlen - 1)//一行的元素 有一点落在1024中就计算
            {
                perfSpB_index row_s = max(perfSpB_index(d_ptr[g_rowid] - i), perfSpB_index(0));
                perfSpB_index row_e = min(perfSpB_index(d_ptr[g_rowid + 1] - i), perfSpB_index(nlen));
                for (perfSpB_index j = row_s + thread_lane; j < row_e; j += THREADS_PER_ROW)
                {
                    sum_tmp += middle_s[j];
                }
                sum_tmp = warpReduceSum<THREADS_PER_ROW>(sum_tmp);
                sum += sum_tmp;
            }
        }
        // Write result
        if (thread_lane == 0)
        {
            d_out[g_rowid] = sum;
        }
    }
}

perfSpB_info perfSpB_SpMV_PCSR_W_FP32(perfSpB_operation_t op,
                                      const void *alpha,
                                      const perfSpB_matrix matA,
                                      const perfSpB_vector vecX,
                                      const void *beta,
                                      perfSpB_vector vecY)
{
    UNUSED(op);
    UNUSED(alpha);
    UNUSED(beta);
    perfSpB_vector_Dense *vecY_csr = (perfSpB_vector_Dense *)(vecY);
    perfSpB_vector_Dense *vecX_csr = (perfSpB_vector_Dense *)(vecX);
    perfSpB_matrix_CSC_or_CSR *matA_csr = (perfSpB_matrix_CSC_or_CSR *)(matA);
    float *d_vecY_csr, *d_vecX_csr, *d_val, *d_v;
    perfSpB_index *d_indices, *d_ptr;

    hipMalloc(&d_vecY_csr, sizeof(float) * vecY->n);
    hipMalloc(&d_vecX_csr, sizeof(float) * vecX->n);
    hipMalloc(&d_val, sizeof(float) * matA_csr->nnz);
    hipMalloc(&d_v, sizeof(float) * matA_csr->nnz);
    hipMalloc(&d_indices, sizeof(perfSpB_index) * matA_csr->nnz);
    hipMalloc(&d_ptr, sizeof(perfSpB_index) * (vecY->n + 1));

    hipMemcpy(d_val, matA_csr->val, sizeof(float) * matA_csr->nnz, hipMemcpyHostToDevice);
    hipMemcpy(d_indices, matA_csr->indices, sizeof(perfSpB_index) * matA_csr->nnz, hipMemcpyHostToDevice);
    hipMemcpy(d_ptr, matA_csr->ptr, sizeof(perfSpB_index) * (vecY->n + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_vecX_csr, ((float *)vecX_csr->values), sizeof(float) * vecX->n, hipMemcpyHostToDevice);
    const int THREADS_PER_ROW = 32;
    const unsigned int ROWS_PER_BLOCK = 8;
    const unsigned int NUM_BLOCKS = static_cast<unsigned int>((vecY->n + (ROWS_PER_BLOCK - 1)) / ROWS_PER_BLOCK);
    cuda_time_test_start();
    for (int i = 0; i < test_iter; i++)
    {
        spmv_pcsr_kernel1<float><<<ceil(matA_csr->nnz / (float)BlockDim), BlockDim>>>(d_val, d_vecX_csr, d_indices, matA_csr->nnz, d_v);
        spmv_pcsr_kernel2<float, ROWS_PER_BLOCK, THREADS_PER_ROW><<<NUM_BLOCKS, threadsPerBlock>>>(d_v, d_ptr, vecY->n, d_vecY_csr);
    }
    cuda_time_test_end();
    double runtime1 = (elapsedTime) / test_iter;
    double gflops = (2.0 * matA_csr->nnz) / ((runtime1 / 1000) * 1e9);

    // cuda_time_test_start();
    // for (int i = 0; i < test_iter; i++)
    // {
    //     // spmv_pcsr_kernel1<float><<<ceil(matA_csr->nnz / (float)BlockDim), BlockDim>>>(d_val, d_vecX_csr, d_indices, matA_csr->nnz, d_v);
    // 	spmv_pcsr_kernel2<float><<<ceil(vecY->n / (float)threadsPerBlock), threadsPerBlock>>>(d_v, d_ptr, vecY->n, d_vecY_csr);
    // }
    // cuda_time_test_end();
    // double runtime2 = (elapsedTime) / test_iter;

    hipDeviceSynchronize();
    printf("\n SpMV CUDA kernel1 runtime = %g ms\n", runtime1);
    // printf("\n SpMV CUDA kernel2 runtime = %g ms\n", runtime2);
    printf("\n SpMV Performance  = %lf GFLOPS\n", gflops);

    hipMemcpy(((float *)vecY_csr->values), d_vecY_csr, sizeof(float) * vecY->n, hipMemcpyDeviceToHost);

    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        printf("Error:%s\n", hipGetErrorString(cudaStatus));
        exit(EXIT_FAILURE);
    }
    hipFree(d_vecY_csr);
    hipFree(d_vecX_csr);
    hipFree(d_val);
    hipFree(d_indices);
    hipFree(d_ptr);

    return perfSpB_success;
}
