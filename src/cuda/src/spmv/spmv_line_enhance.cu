#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <spmv_cuda.h>
#include <math.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <algorithm>  // 包含 std::sort
#include <functional> // 包含 std::greater

#include "cuda_timer.h"

#define test_iter 1


template <typename I, typename T>
__device__ __forceinline__ void line_enhance_direct_reduce(const I reduce_row_id, const I block_row_end,
                                                           const I reduce_row_idx_begin, const I reduce_row_idx_end,
                                                           const I block_round_inx_start, const I block_round_inx_end,
                                                           const T *shared_val, T &sum) {
  if (reduce_row_id < block_row_end) {
    if (reduce_row_idx_begin < block_round_inx_end && reduce_row_idx_end > block_round_inx_start) {//reduce_range ← vec_range ∩ round_range
      const I reduce_start = max(reduce_row_idx_begin, block_round_inx_start);
      const I reduce_end = min(reduce_row_idx_end, block_round_inx_end);
      for (I j = reduce_start; j < reduce_end; j++) {
        sum += shared_val[j - block_round_inx_start];
      }
    }
  }
}

template <perfSpB_index R, int VEC_SIZE, int ROWS_PER_BLOCK, perfSpB_index THREADS_PER_BLOCK, typename I, typename T>
__global__ void spmv_line_enhance_kernel(T *d_val,
                                         perfSpB_index *d_ptr,
                                         perfSpB_index *d_cols,
                                         perfSpB_index d_nnz,
                                         perfSpB_index nrows,
                                         T *d_vector,
                                         T *d_out)
{
  static_assert(THREADS_PER_BLOCK / VEC_SIZE >= ROWS_PER_BLOCK,
                "vector number in block must larger or equal then the rows processed per block");

  const int g_tid = threadIdx.x + blockDim.x * blockIdx.x; // global thread id
  const int g_bid = blockIdx.x;                            // global block id
  const int tid_in_block = g_tid % THREADS_PER_BLOCK;      // local thread id in current block

  const int shared_len = THREADS_PER_BLOCK * R;
  __shared__ T shared_val[shared_len];

  const I block_row_begin = g_bid * ROWS_PER_BLOCK; // 一个线程块的起始行
  const I block_row_end = min(block_row_begin + ROWS_PER_BLOCK, nrows);

  const I block_row_idx_start = d_ptr[block_row_begin];
  const I block_row_idx_end = d_ptr[block_row_end]; // 此时会触发访存合并，由于同一线程块访问同一地址。比第一个线程加载然后广播性能要高

  //////////////////////////////////////////////////////////////////////////////
  const I vec_id_in_block = g_tid / VEC_SIZE % (THREADS_PER_BLOCK / VEC_SIZE);
  const I tid_in_vec = g_tid % VEC_SIZE;
  const I reduce_row_id = block_row_begin + vec_id_in_block;
  I reduce_row_idx_begin = 0;
  I reduce_row_idx_end = 0;
  if (reduce_row_id < block_row_end)
  {
    reduce_row_idx_begin = d_ptr[reduce_row_id];
    reduce_row_idx_end = d_ptr[reduce_row_id + 1];
  }
  //////////////////////////////////////////////////////////////////////////////

  T sum = static_cast<T>(0);
  const int rounds =
      (block_row_idx_end - block_row_idx_start) / (R * THREADS_PER_BLOCK) +
      ((block_row_idx_end - block_row_idx_start) % (R * THREADS_PER_BLOCK) == 0 ? 0 : 1); // 一个round加载(R * THREADS)个元素

  for (int r = 0; r < rounds; r++)
  {
    // start and end data index in each round
    const I block_round_inx_start = block_row_idx_start + r * R * THREADS_PER_BLOCK;
    const I block_round_inx_end = min(block_round_inx_start + R * THREADS_PER_BLOCK, block_row_idx_end);
    I i = block_round_inx_start + tid_in_block;

    __syncthreads();
// in each inner loop, it processes R*THREADS element at max
#pragma unroll
    for (int k = 0; k < R; k++)
    { // streaming
      if (i < block_row_idx_end)
      {
        const T tmp = d_val[i] * d_vector[d_cols[i]];
        shared_val[i - block_round_inx_start] = tmp;
      }
      i += THREADS_PER_BLOCK;
    }
    __syncthreads();
    // reduce
    {
      line_enhance_direct_reduce<I, T>(reduce_row_id, block_row_end, reduce_row_idx_begin, reduce_row_idx_end,
                                       block_round_inx_start, block_round_inx_end, shared_val, sum);
    }

    if (reduce_row_id < block_row_end)
    {
      d_out[reduce_row_id] = sum + d_out[reduce_row_id];
    }
  }

}

perfSpB_info perfSpB_SpMV_LINE_ENHANCE_FP32(perfSpB_operation_t op,
                                            const void *alpha,
                                            const perfSpB_matrix matA,
                                            const perfSpB_vector vecX,
                                            const void *beta,
                                            perfSpB_vector vecY)
{
  UNUSED(op);
  UNUSED(alpha);
  UNUSED(beta);
  perfSpB_vector_Dense *vecY_csr = (perfSpB_vector_Dense *)(vecY);
  perfSpB_vector_Dense *vecX_csr = (perfSpB_vector_Dense *)(vecX);
  perfSpB_matrix_CSC_or_CSR *matA_csr = (perfSpB_matrix_CSC_or_CSR *)(matA);
  perfSpB_index nrows = matA->row;
  // perfSpB_index ncols = matA->col;
  perfSpB_index nvals = matA_csr->nnz;

  float *d_vecY_csr, *d_vecX_csr, *d_val;
  perfSpB_index *d_indices, *d_ptr;
  for (int i = 0; i < matA->row; i++)
  {
    if (*((float *)vecY_csr->values) != 0)
    {
      printf("\n %f \n", *((float *)vecY_csr->values));
    }
  }

  hipMalloc(&d_vecY_csr, sizeof(float) * vecY->n);
  hipMalloc(&d_vecX_csr, sizeof(float) * vecX->n);
  hipMalloc(&d_val, sizeof(float) * matA_csr->nnz);
  hipMalloc(&d_indices, sizeof(perfSpB_index) * matA_csr->nnz);
  hipMalloc(&d_ptr, sizeof(perfSpB_index) * (vecY->n + 1));

  hipMemcpy(d_val, matA_csr->val, sizeof(float) * matA_csr->nnz, hipMemcpyHostToDevice);
  hipMemcpy(d_indices, matA_csr->indices, sizeof(perfSpB_index) * matA_csr->nnz, hipMemcpyHostToDevice);
  hipMemcpy(d_ptr, matA_csr->ptr, sizeof(perfSpB_index) * (vecY->n + 1), hipMemcpyHostToDevice);
  hipMemcpy(d_vecX_csr, ((float *)vecX_csr->values), sizeof(float) * vecX->n, hipMemcpyHostToDevice);
  hipMemcpy(d_vecY_csr, ((float *)vecY_csr->values), sizeof(float) * vecY->n, hipMemcpyHostToDevice);

  const int bp_1 = matA_csr->ptr[nrows / 2];
  const int bp_2 = matA_csr->ptr[nrows];

  const int nnz_block_0 = bp_1 - 0;
  const int nnz_block_1 = bp_2 - bp_1;
  int avg_block_nnz_max = max(2 * nnz_block_0 / nrows, 2 * nnz_block_1 / nrows);

  const int R = 2;
  const int THREADS_PER_BLOCK = 512;
  constexpr int ROWS_PER_BLOCK = 32;

  const int WORK_BLOCKS = (nrows / ROWS_PER_BLOCK) + ((nrows % ROWS_PER_BLOCK == 0) ? 0 : 1);

  cuda_time_test_start();
  for (int i = 0; i < test_iter; i++)
  {
    spmv_line_enhance_kernel<R, 4, ROWS_PER_BLOCK, THREADS_PER_BLOCK, perfSpB_index, float><<<(WORK_BLOCKS), (THREADS_PER_BLOCK)>>>(d_val, d_ptr, d_indices, nvals, nrows, d_vecX_csr, d_vecY_csr);
  }
  cuda_time_test_end();
  double runtime = (elapsedTime) / test_iter;
  double gflops = (2.0 * matA_csr->nnz) / ((runtime / 1000) * 1e9);

  hipDeviceSynchronize();
  printf("\n SpMV CUDA kernel runtime = %g ms\n", runtime);
  // printf("\n SpMV CUDA kernel2 runtime = %g ms\n", runtime2);
  printf("\n SpMV Performance  = %lf GFLOPS\n", gflops);

  hipMemcpy(((float *)vecY_csr->values), d_vecY_csr, sizeof(float) * vecY->n, hipMemcpyDeviceToHost);

  hipError_t cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess)
  {
    printf("Error:%s\n", hipGetErrorString(cudaStatus));
    exit(EXIT_FAILURE);
  }

  hipFree(d_vecY_csr);
  hipFree(d_vecX_csr);
  hipFree(d_val);
  hipFree(d_indices);
  hipFree(d_ptr);

  return perfSpB_success;
}
