#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <spmv_cuda.h>
#include <math.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>

#include "cuda_timer.h"

#define test_iter 2000

#define warpSize 32

#define BlockDim 1024
#define BLOCKS_EXP 640
#define ITER 3

#define SHARED_X_PER_BLOCK 3072
#define NUM_THREAD_PER_BLOCK 256
#define num_loads_per_thread 12

// calculates the number of rows of a CSR matrix that can fit into BlockDim entries.
int spmv_csr_adaptive_rowblocks(perfSpB_index *ptr, int totalRows, int *rowBlocks, perfSpB_index nnzPerBlock)
{
    rowBlocks[0] = 0;
    int sum = 0;
    int last_i = 0;
    int ctr = 1;
    for (int i = 1; i < totalRows; i++)
    {
        // Count non-zeroes in this row
        sum += ptr[i] - ptr[i - 1];
        if (sum == nnzPerBlock)
        {
            // This row fills up LOCAL_SIZE
            last_i = i;
            rowBlocks[ctr++] = i;
            sum = 0;
        }
        else if (sum > nnzPerBlock)
        {
            if (i - last_i > 1)
            {
                // This extra row will not fit
                rowBlocks[ctr++] = i - 1;
                i--;
            }
            else if (i - last_i == 1)
                // This one row is too large
                rowBlocks[ctr++] = i;
            last_i = i;
            sum = 0;
        }
    }
    rowBlocks[ctr++] = totalRows;
    return ctr;
}

template <unsigned int threads_per_row>
__device__ __forceinline__ float warpReduceSum(float sum)
{
    if (threads_per_row >= 32)
        sum += __shfl_down_sync(0xffffffff, sum, 16); // 0-16, 1-17, 2-18, etc.
    if (threads_per_row >= 16)
        sum += __shfl_down_sync(0xffffffff, sum, 8); // 0-8, 1-9, 2-10, etc.
    if (threads_per_row >= 8)
        sum += __shfl_down_sync(0xffffffff, sum, 4); // 0-4, 1-5, 2-6, etc.
    if (threads_per_row >= 4)
        sum += __shfl_down_sync(0xffffffff, sum, 2); // 0-2, 1-3, 4-6, 5-7, etc.
    if (threads_per_row >= 2)
        sum += __shfl_down_sync(0xffffffff, sum, 1); // 0-1, 2-3, 4-5, etc.
    return sum;
}
template <typename ValueType, unsigned int VECTORS_PER_BLOCK, unsigned int THREADS_PER_VECTOR>
__global__ void nec_spmv_v(ValueType *__restrict__ y,
                         const ValueType *__restrict__ A_value,
                         const perfSpB_index *__restrict__ A_row_offset,
                         int * __restrict__ A_ecr_indices,
                         ValueType *__restrict__ ecr_x,
                         perfSpB_index row_num,
                         int *__restrict__ ROW_BLOCK,
                         int *__restrict__ row_block_ptr)
{
    float *local_ecr_xx_val = ecr_x + row_block_ptr[blockIdx.x];
    __shared__ volatile float x_cache[SHARED_X_PER_BLOCK];

    for (int i = 0; i < num_loads_per_thread; i++)
    {
        unsigned int cache_idx = i * NUM_THREAD_PER_BLOCK + threadIdx.x;
        if (cache_idx < SHARED_X_PER_BLOCK)
            x_cache[cache_idx] = local_ecr_xx_val[cache_idx];
    }
    __syncthreads();

    // const perfSpB_index thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    const perfSpB_index tid_in_vec = threadIdx.x & (THREADS_PER_VECTOR - 1);
    const perfSpB_index vec_id = threadIdx.x / THREADS_PER_VECTOR;

    const perfSpB_index reduce_start_row_id = ROW_BLOCK[blockIdx.x];
    const perfSpB_index reduce_end_row_id = ROW_BLOCK[blockIdx.x + 1];
    // int num_rows = reduce_end_row_id - reduce_start_row_id;
    int reduce_row_id = reduce_start_row_id + vec_id;

    for (; reduce_row_id < reduce_end_row_id; reduce_row_id += VECTORS_PER_BLOCK)
    {
        float sum = static_cast<float>(0);
        for (int jj = A_row_offset[reduce_row_id] + tid_in_vec; jj < A_row_offset[reduce_row_id + 1]; jj += THREADS_PER_VECTOR)
        {
            int col = A_ecr_indices[jj];
            if(col < SHARED_X_PER_BLOCK)
            {
                sum += A_value[jj] * x_cache[col];
            }
            else
            {
                sum += A_value[jj] * local_ecr_xx_val[col];
            }
        }
        sum = warpReduceSum<THREADS_PER_VECTOR>(sum);
        // store value
        if (tid_in_vec == 0)
        {
            // atomicAdd(y + reduce_row_id, sum);
            y[reduce_row_id] = sum;
        }
    }
}



// __global__ void nec_spmv(float *__restrict__ y,
//                          const float *__restrict__ A_value,
//                          const perfSpB_index *__restrict__ A_row_offset,
//                          int * __restrict__ A_ecr_indices,
//                          float *__restrict__ ecr_x,
//                          perfSpB_index row_num,
//                          int *__restrict__ ROW_BLOCK,
//                          int *__restrict__ row_block_ptr)
// {
//     float *local_ecr_xx_val = ecr_x + row_block_ptr[blockIdx.x];
//     __shared__ volatile float x_cache[SHARED_X_PER_BLOCK];

//     for (int i = 0; i < num_loads_per_thread; i++)
//     {
//         unsigned int cache_idx = i * NUM_THREAD_PER_BLOCK + threadIdx.x;
//         if (cache_idx < SHARED_X_PER_BLOCK)
//             x_cache[cache_idx] = local_ecr_xx_val[cache_idx];
//     }
//     __syncthreads();

//     const perfSpB_index reduce_start_row_id = ROW_BLOCK[blockIdx.x];
//     const perfSpB_index reduce_end_row_id = ROW_BLOCK[blockIdx.x + 1];
//     // int num_rows = reduce_end_row_id - reduce_start_row_id;
//     int reduce_row_id = reduce_start_row_id + threadIdx.x;

//     for (; reduce_row_id < reduce_end_row_id; reduce_row_id += blockDim.x)
//     {
//         float sum = static_cast<float>(0);
//         for (int jj = A_row_offset[reduce_row_id]; jj < A_row_offset[reduce_row_id + 1]; jj ++)
//         {
//             int col = A_ecr_indices[jj];
//             if(col < SHARED_X_PER_BLOCK)
//             {
//                 sum += A_value[jj] * x_cache[col];
//             }
//             else
//             {
//                 sum += A_value[jj] * local_ecr_xx_val[col];
//             }
//         }
//         y[reduce_row_id] = sum;
//     }
// }

template <typename ValueType>
__global__ void nec_vector_preprocess_kernel(ValueType *ecr_x,
                                             int *use_x_indices,
                                             const ValueType *x,
                                             int ecr_x_size)
{
    int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    if (thread_id < ecr_x_size)
    {
        ecr_x[thread_id] = *((ValueType *)x + use_x_indices[thread_id]);
    }
}

perfSpB_info perfSpB_SpMV_Nec_FP32(perfSpB_operation_t op,
                                   const void *alpha,
                                   const perfSpB_matrix matA,
                                   const perfSpB_vector vecX,
                                   const void *beta,
                                   perfSpB_vector vecY)
{
    UNUSED(op);
    UNUSED(alpha);
    UNUSED(beta);
    perfSpB_vector_Dense *vecY_csr = (perfSpB_vector_Dense *)(vecY);
    perfSpB_vector_Dense *vecX_csr = (perfSpB_vector_Dense *)(vecX);
    perfSpB_matrix_CSC_or_CSR *matA_csr = (perfSpB_matrix_CSC_or_CSR *)(matA);
    perfSpB_index nvals = matA_csr->nnz;
    perfSpB_index nrows = matA->row;
    perfSpB_index ncols = matA->col;

    perfSpB_index NNZ_PER_BLOCK = ceil(nvals / (BLOCKS_EXP));

    int *rowBlocks, *d_rowBlocks;
    rowBlocks = (int *)malloc((vecY->n) * sizeof(int));
    int countRowBlocks = spmv_csr_adaptive_rowblocks(matA_csr->ptr, vecY->n, rowBlocks, NNZ_PER_BLOCK);
    printf("\n countRowBlocks = %d \n", countRowBlocks);
    //////////////////////////////////ECR start//////////////////////////////////
    int *NEC_NUM;
    int **use_x_indices;
    float **ecr_xx_val;

    perfSpB_index A_COLS = ncols;
    perfSpB_index A_ROWS = nrows;
    int *ecr_indices = (int *)malloc((nvals + 10) * sizeof(int));
    memset(ecr_indices, 0, (nvals + 10) * sizeof(int));

    NEC_NUM = (int *)malloc((countRowBlocks + 1) * sizeof(int));
    memset(NEC_NUM, 0, sizeof(int) * (countRowBlocks + 1));

    use_x_indices = (int **)malloc((countRowBlocks + 1) * sizeof(int *));
    ecr_xx_val = (float **)malloc((countRowBlocks + 1) * sizeof(float *));
    int acc_nec_num = 0;
    for (int row_blk_id = 0; row_blk_id < countRowBlocks; row_blk_id++)
    {
        int *not_null_col_flag = (int *)malloc(A_COLS * sizeof(int));
        int *IDX_MAP = (int *)malloc(A_COLS * sizeof(int));
        int *IDX_OFFSET = (int *)malloc(A_COLS * sizeof(int));
        use_x_indices[row_blk_id] = (int *)malloc(A_COLS * sizeof(int));
        memset(use_x_indices[row_blk_id], 0, A_COLS * sizeof(int));

        int startRow = rowBlocks[row_blk_id];
        int nextStartRow = rowBlocks[row_blk_id + 1];

        for (perfSpB_index col = 0; col < A_COLS; col++)
        {
            IDX_MAP[col] = col;
            not_null_col_flag[col] = 1;
            IDX_OFFSET[col] = 1;
        }
        for (int j = matA_csr->ptr[startRow]; j < matA_csr->ptr[nextStartRow]; ++j)
        {
            not_null_col_flag[matA_csr->indices[j]] = 0;
        }
        IDX_OFFSET[0] = not_null_col_flag[0];
        for (perfSpB_index col = 1; col < A_COLS; col++)
        {
            IDX_OFFSET[col] = IDX_OFFSET[col - 1] + not_null_col_flag[col];
        }
        for (perfSpB_index col = 0; col < A_COLS; col++)
        {
            IDX_MAP[col] = IDX_MAP[col] - IDX_OFFSET[col];
        }
        {
            for (int j = matA_csr->ptr[startRow]; j < matA_csr->ptr[nextStartRow]; ++j)
            {
                ecr_indices[j] = IDX_MAP[matA_csr->indices[j]];
            }
        }
        int p = 0;
        for (perfSpB_index col = 0; col < A_COLS; col++)
        {
            if (not_null_col_flag[col] != 1)
            {
                use_x_indices[row_blk_id][p] = col;
                p++;
            }
        }
        NEC_NUM[row_blk_id] = p;
        acc_nec_num += p;
        ecr_xx_val[row_blk_id] = (float *)malloc(p * sizeof(float));
        free(not_null_col_flag);
        free(IDX_MAP);
        free(IDX_OFFSET);
    }
    // use_x_indices 转换为一维 h_use_x_indices
    int *row_blk_ptr = (int *)malloc((countRowBlocks) * sizeof(int));
    int *h_use_x_indices = (int *)malloc((acc_nec_num) * sizeof(int));
    float *h_ecr_xx_val = (float *)malloc((acc_nec_num) * sizeof(float));
    memset(h_use_x_indices, 0, (acc_nec_num) * sizeof(int));
    memset(h_ecr_xx_val, 0.0, (acc_nec_num) * sizeof(float));
    perfSpB_index p = 0, acc = 0;
    for (int row_blk_id = 0; row_blk_id < countRowBlocks; row_blk_id++)
    {
        int nec_num = NEC_NUM[row_blk_id];
        row_blk_ptr[row_blk_id] = acc;
        for (int j = 0; j < nec_num; j++)
        {
            h_use_x_indices[p++] = use_x_indices[row_blk_id][j];
        }
        acc += nec_num;
    }

    // row 与  每个行块id的映射
    perfSpB_index *h_ROW_BLOCK_ID = (perfSpB_index *)malloc((A_ROWS) * sizeof(perfSpB_index));
    memset(h_ROW_BLOCK_ID, 0, (A_ROWS) * sizeof(perfSpB_index));
    for (int row_blk_id = 0; row_blk_id < countRowBlocks; row_blk_id++)
    {
        for (int row = rowBlocks[row_blk_id]; row < rowBlocks[row_blk_id + 1]; ++row)
        {
            h_ROW_BLOCK_ID[row] = row_blk_id;
        }
    }

    //////////////////////////////////ECR  end//////////////////////////////////
    int *d_NEC_NUM;
    int *d_use_x_indices;
    int *d_row_blk_ptr;
    float *d_ecr_xx_val;
    float *d_vecX_csr;

    hipMalloc(&d_NEC_NUM, (countRowBlocks + 1) * sizeof(int));
    hipMalloc(&d_use_x_indices, acc_nec_num * sizeof(int));
    hipMalloc(&d_row_blk_ptr, countRowBlocks * sizeof(int));
    hipMalloc(&d_ecr_xx_val, acc_nec_num * sizeof(float));
    hipMalloc(&d_vecX_csr, sizeof(float) * vecX->n);
    hipMemcpy(d_NEC_NUM, NEC_NUM, sizeof(int) * (countRowBlocks + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_use_x_indices, h_use_x_indices, sizeof(int) * acc_nec_num, hipMemcpyHostToDevice);
    hipMemcpy(d_row_blk_ptr, row_blk_ptr, sizeof(int) * countRowBlocks, hipMemcpyHostToDevice);
    hipMemcpy(d_ecr_xx_val, h_ecr_xx_val, sizeof(float) * acc_nec_num, hipMemcpyHostToDevice);
    hipMemcpy(d_vecX_csr, ((float *)vecX_csr->values), sizeof(float) * vecX->n, hipMemcpyHostToDevice);

    float *d_vecY_csr, *d_val;
    perfSpB_index *d_ptr, *d_ROW_BLOCK_ID;
    int *d_ecr_indices;
    hipMalloc(&d_vecY_csr, sizeof(float) * vecY->n);
    hipMalloc(&d_val, sizeof(float) * matA_csr->nnz);
    hipMalloc(&d_ptr, sizeof(perfSpB_index) * (vecY->n + 1));
    hipMalloc(&d_ROW_BLOCK_ID, sizeof(perfSpB_index) * A_ROWS);
    hipMalloc(&d_ecr_indices, (nvals + 10) * sizeof(int));
    hipMemcpy(d_val, matA_csr->val, sizeof(float) * matA_csr->nnz, hipMemcpyHostToDevice);
    hipMemcpy(d_ptr, matA_csr->ptr, sizeof(perfSpB_index) * (vecY->n + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_ROW_BLOCK_ID, h_ROW_BLOCK_ID, sizeof(perfSpB_index) * A_ROWS, hipMemcpyHostToDevice);
    hipMemcpy(d_ecr_indices, ecr_indices, sizeof(int) * (nvals + 10), hipMemcpyHostToDevice);

    hipMalloc(&d_rowBlocks, countRowBlocks * sizeof(int));
    hipMemcpy(d_rowBlocks, rowBlocks, countRowBlocks * sizeof(int), hipMemcpyHostToDevice);

    unsigned int NUM_BLOCKS = static_cast<unsigned int>((acc_nec_num + (1024 - 1)) / 1024);
    
    cuda_time_test_start();
    for (int i = 0; i < test_iter; i++)
    {
        nec_vector_preprocess_kernel<float><<<NUM_BLOCKS, 1024>>>(d_ecr_xx_val, d_use_x_indices, d_vecX_csr, acc_nec_num);
        nec_spmv_v<float, 128, 2><<<countRowBlocks, 256>>>(d_vecY_csr, d_val, d_ptr, d_ecr_indices, d_ecr_xx_val, vecY->n, d_rowBlocks, d_row_blk_ptr);
        // nec_spmv<<<countRowBlocks, 512>>>(d_vecY_csr, d_val, d_ptr, d_ecr_indices, d_ecr_xx_val, vecY->n, d_rowBlocks, d_row_blk_ptr);
    }

    cuda_time_test_end();
    double runtime1 = (elapsedTime) / test_iter;
    double gflops = (2.0 * matA_csr->nnz) / ((runtime1 / 1000) * 1e9);

    hipDeviceSynchronize();
    printf("\n ------------------------------------------------------------------------------");
    printf("\n SpMV CUDA kernel runtime = %g ms\n", runtime1);
    // printf("\n SpMV CUDA kernel2 runtime = %g ms\n", runtime2);
    printf("\n SpMV Performance  = %lf GFLOPS\n", gflops);
    printf(" ------------------------------------------------------------------------------ \n");

    hipMemcpy(((float *)vecY_csr->values), d_vecY_csr, sizeof(float) * vecY->n, hipMemcpyDeviceToHost);

    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        printf("Error:%s\n", hipGetErrorString(cudaStatus));
        exit(EXIT_FAILURE);
    }
    hipFree(d_vecY_csr);
    hipFree(d_vecX_csr);
    hipFree(d_val);
    hipFree(d_ptr);

    return perfSpB_success;
}
