#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <spmv_cuda.h>
#include "cuda_timer.h"

#define test_iter 2000
#define NUM_ROW_BLOCKS 128

void MergePathDivide(
    int diagonal,
    perfSpB_index *a,
    int *b,
    int a_len,
    int b_len,
    perfSpB_coord *path_coordinate)
{
    // Diagonal search range (in x coordinate space)
    int x_min = max(diagonal - b_len, 0);
    int x_max = min(diagonal, a_len);
    // printf("\n x_min = %d and x_max = %d \n", x_min, x_max);

    // 2D binary-search along the diagonal search range
    while (x_min < x_max)
    {
        int pivot = (x_min + x_max) >> 1;
        if ((int)a[pivot] <= b[diagonal - pivot - 1])
        {
            // Keep top-right half of diagonal range
            x_min = pivot + 1;
        }
        else
        {
            // Keep bottom-left half of diagonal range
            x_max = pivot;
        }
    }
    path_coordinate->x = min(x_min, a_len);
    path_coordinate->y = diagonal - x_min;
}

template <unsigned int threads_per_row>
__device__ __forceinline__ float warpReduceSum(float sum)
{
    if (threads_per_row >= 32)
        sum += __shfl_down_sync(0xffffffff, sum, 16); // 0-16, 1-17, 2-18, etc.
    if (threads_per_row >= 16)
        sum += __shfl_down_sync(0xffffffff, sum, 8); // 0-8, 1-9, 2-10, etc.
    if (threads_per_row >= 8)
        sum += __shfl_down_sync(0xffffffff, sum, 4); // 0-4, 1-5, 2-6, etc.
    if (threads_per_row >= 4)
        sum += __shfl_down_sync(0xffffffff, sum, 2); // 0-2, 1-3, 4-6, 5-7, etc.
    if (threads_per_row >= 2)
        sum += __shfl_down_sync(0xffffffff, sum, 1); // 0-1, 2-3, 4-5, etc.
    return sum;
}

// template <typename ValueType, unsigned int VECTORS_PER_BLOCK, unsigned int THREADS_PER_VECTOR>
// __global__ void spmv_vnec_kernel_cuda(ValueType *__restrict__ y,
//                                       const ValueType *__restrict__ A_value,
//                                       const perfSpB_index *__restrict__ A_col_index,
//                                       const perfSpB_index *__restrict__ A_row_offset,
//                                       const ValueType *__restrict__ x,
//                                       perfSpB_index row_num)
// {
//     const perfSpB_index THREADS_PER_BLOCK = VECTORS_PER_BLOCK * THREADS_PER_VECTOR;
//     const perfSpB_index thread_id = THREADS_PER_BLOCK * blockIdx.x + threadIdx.x; // global thread index
//     const perfSpB_index thread_lane = threadIdx.x & (THREADS_PER_VECTOR - 1);     // thread index within the vector/row
//     const perfSpB_index row_id = thread_id / THREADS_PER_VECTOR;                  // global vector index

//     if (row_id < row_num)
//     {
//         const perfSpB_index row_start = A_row_offset[row_id]; // same as: row_start = Ap[row];
//         const perfSpB_index row_end = A_row_offset[row_id + 1];

//         // initialize local sum
//         ValueType sum = 0;

//         // accumulate local sums
//         for (perfSpB_index jj = row_start + thread_lane; jj < row_end; jj += THREADS_PER_VECTOR)
//             sum += A_value[jj] * x[A_col_index[jj]];

//         sum = warpReduceSum<THREADS_PER_VECTOR>(sum);
//         if (thread_lane == 0)
//         {
//             y[row_id] = sum;
//         }
//     }
// }



template <typename ValueType, unsigned int VECTORS_PER_BLOCK, unsigned int THREADS_PER_VECTOR>
__global__ void vnec_spmv(ValueType *__restrict__ y,
                                      const ValueType *__restrict__ A_value,
                                      const perfSpB_index *__restrict__ A_row_offset,
                                      int * A_ecr_indices,
                                      ValueType * ecr_x,
                                      perfSpB_index row_num,
                                      perfSpB_index *ROW_BLOCK_ID,
                                      int * row_block_ptr)
{
    const perfSpB_index thread_id = blockDim.x * blockIdx.x + threadIdx.x; // global thread index
    const perfSpB_index thread_lane = threadIdx.x & (THREADS_PER_VECTOR - 1);     // thread index within the vector/row
    const perfSpB_index row_id = thread_id / THREADS_PER_VECTOR;                  // global vector index
    const perfSpB_index row_block_id = ROW_BLOCK_ID[row_id];                  // global vector index

    perfSpB_index row_block_start = row_block_ptr[row_block_id];
    float *local_ecr_xx_val = ecr_x + row_block_start;

    if (row_id < row_num)
    {
        const perfSpB_index row_start = A_row_offset[row_id]; // same as: row_start = Ap[row];
        const perfSpB_index row_end = A_row_offset[row_id + 1];

        // initialize local sum
        ValueType sum = 0;

        // accumulate local sums
        for (perfSpB_index jj = row_start + thread_lane; jj < row_end; jj += THREADS_PER_VECTOR)
            sum += A_value[jj] * local_ecr_xx_val[A_ecr_indices[jj]];

        sum = warpReduceSum<THREADS_PER_VECTOR>(sum);
        if (thread_lane == 0)
        {
            y[row_id] = sum;
        }
    }
}


template <typename ValueType, unsigned int VECTORS_PER_BLOCK, unsigned int THREADS_PER_VECTOR>
__global__ void vnec_spmv_L(ValueType *__restrict__ y,
                                      const ValueType *__restrict__ A_value,
                                      const perfSpB_index *__restrict__ A_row_offset,
                                      int * A_ecr_indices,
                                      ValueType * ecr_x,
                                      perfSpB_index row_num,
                                      perfSpB_index *ROW_BLOCK_ID,
                                      int * row_block_ptr)
{
    const perfSpB_index thread_id = blockDim.x * blockIdx.x + threadIdx.x; // global thread index
    const perfSpB_index thread_lane = threadIdx.x & (THREADS_PER_VECTOR - 1);     // thread index within the vector/row
    const perfSpB_index row_id = thread_id / THREADS_PER_VECTOR;                  // global vector index
    const perfSpB_index row_block_id = ROW_BLOCK_ID[row_id];                  // global vector index
    const perfSpB_index lane_id = threadIdx.x % 32;                         // global vector index

    perfSpB_index row_block_start = row_block_ptr[row_block_id];
    float *local_ecr_xx_val = ecr_x + row_block_start;

    if (row_id < row_num)
    {
        const perfSpB_index row_start = A_row_offset[row_id]; // same as: row_start = Ap[row];
        const perfSpB_index row_end = A_row_offset[row_id + 1];

        // initialize local sum
        ValueType sum = 0;

        // accumulate local sums
        for (perfSpB_index jj = row_start + thread_lane; jj < row_end; jj += THREADS_PER_VECTOR)
            sum += A_value[jj] * local_ecr_xx_val[A_ecr_indices[jj]];

        for (int offset = 16; offset > 0; offset >>= 1)
        {
            sum += __shfl_down_sync(0xffffffff, sum, offset);
        }
        if (lane_id == 0)
        {
            atomicAdd(&y[row_id], sum);
        }
    }
}

// TODO: 时间太长
template <typename ValueType>
__global__ void vector_preprocess_kernel(ValueType *ecr_x,
                        int *use_x_indices,
                        const ValueType *x,
                        int ecr_x_size)
{
    int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    if(thread_id < ecr_x_size)
    {
        ecr_x[thread_id] = *((ValueType *)x + use_x_indices[thread_id]);
    }
}

perfSpB_info perfSpB_SpMV_VNEC_FP32(perfSpB_operation_t op,
                                    const void *alpha,
                                    const perfSpB_matrix matA,
                                    const perfSpB_vector vecX,
                                    const void *beta,
                                    perfSpB_vector vecY)
{
    UNUSED(op);
    UNUSED(alpha);
    UNUSED(beta);
    perfSpB_vector_Dense *vecY_csr = (perfSpB_vector_Dense *)(vecY);
    perfSpB_vector_Dense *vecX_csr = (perfSpB_vector_Dense *)(vecX);
    perfSpB_matrix_CSC_or_CSR *matA_csr = (perfSpB_matrix_CSC_or_CSR *)(matA);

    int *NEC_NUM;
    int **use_x_indices;
    float **ecr_xx_val;

    perfSpB_index nrows = matA->row;
    perfSpB_index ncols = matA->col;
    perfSpB_index nvals = matA_csr->nnz;
    /////////////////////////////////////merge-path opt/////////////////////////////////////
    int num_merge_items = nvals + nrows;
    int items_per_thread = (num_merge_items + NUM_ROW_BLOCKS - 1) / NUM_ROW_BLOCKS;
    int *diagonal_start = (int *)malloc((NUM_ROW_BLOCKS) * sizeof(int));
    int *diagonal_end = (int *)malloc((NUM_ROW_BLOCKS) * sizeof(int));
    perfSpB_coord *row_block_coo_start = (perfSpB_coord *)malloc((NUM_ROW_BLOCKS) * sizeof(perfSpB_coord));
    perfSpB_coord *row_block_coo_end = (perfSpB_coord *)malloc((NUM_ROW_BLOCKS) * sizeof(perfSpB_coord));
    int *nz_indices = (int *)malloc((nvals) * sizeof(int));
    for (perfSpB_index i = 0; i < nvals; i++)
    {
        nz_indices[i] = i;
    }
    for (int row_blk_id = 0; row_blk_id < NUM_ROW_BLOCKS; row_blk_id++)
    {
        diagonal_start[row_blk_id] = min(items_per_thread * row_blk_id, num_merge_items);
        diagonal_end[row_blk_id] = min(diagonal_start[row_blk_id] + items_per_thread, num_merge_items);
        MergePathDivide(diagonal_start[row_blk_id], (matA_csr->ptr + 1), nz_indices, nrows, nvals, row_block_coo_start + row_blk_id);
        MergePathDivide(diagonal_end[row_blk_id], (matA_csr->ptr + 1), nz_indices, nrows, nvals, row_block_coo_end + row_blk_id);
        row_block_coo_end[row_blk_id].x = (row_block_coo_end[row_blk_id].x) + 1;
        row_block_coo_start[row_blk_id].x = (row_block_coo_start[row_blk_id].x) + 1;
        if (row_blk_id == 0)
            row_block_coo_start[row_blk_id].x--;
        if (row_blk_id == NUM_ROW_BLOCKS - 1)
            row_block_coo_end[row_blk_id].x--;
        row_block_coo_start[row_blk_id].y = matA_csr->ptr[row_block_coo_start[row_blk_id].x];
        row_block_coo_end[row_blk_id].y = matA_csr->ptr[row_block_coo_end[row_blk_id].x];
    }
    /////////////////////////////////////ECR/////////////////////////////////////

    perfSpB_index A_COLS = ncols;
    perfSpB_index A_ROWS = nrows;
    int *ecr_indices = (int *)malloc((nvals + 10) * sizeof(int));
    memset(ecr_indices, 0, (nvals + 10) * sizeof(int));

    NEC_NUM = (int *)malloc((NUM_ROW_BLOCKS + 1) * sizeof(int));
    memset(NEC_NUM, 0, sizeof(int) * (NUM_ROW_BLOCKS + 1));

    use_x_indices = (int **)malloc((NUM_ROW_BLOCKS + 1) * sizeof(int *));
    ecr_xx_val = (float **)malloc((NUM_ROW_BLOCKS + 1) * sizeof(float *));
    int acc_nec_num = 0;
    for (int row_blk_id = 0; row_blk_id < NUM_ROW_BLOCKS; row_blk_id++)
    {
        int *not_null_col_flag = (int *)malloc(A_COLS * sizeof(int));
        int *IDX_MAP = (int *)malloc(A_COLS * sizeof(int));
        int *IDX_OFFSET = (int *)malloc(A_COLS * sizeof(int));
        use_x_indices[row_blk_id] = (int *)malloc(A_COLS * sizeof(int));
        memset(use_x_indices[row_blk_id], 0, A_COLS * sizeof(int));

        perfSpB_coord row_block_coo_start_t = row_block_coo_start[row_blk_id];
        perfSpB_coord row_block_coo_end_t = row_block_coo_end[row_blk_id];

        for (perfSpB_index col = 0; col < A_COLS; col++)
        {
            IDX_MAP[col] = col;
            not_null_col_flag[col] = 1;
            IDX_OFFSET[col] = 1;
        }
        for (int j = row_block_coo_start_t.y; j < row_block_coo_end_t.y; ++j)
        {
            not_null_col_flag[matA_csr->indices[j]] = 0;
        }
        IDX_OFFSET[0] = not_null_col_flag[0];
        for (perfSpB_index col = 1; col < A_COLS; col++)
        {
            IDX_OFFSET[col] = IDX_OFFSET[col - 1] + not_null_col_flag[col];
        }
        for (perfSpB_index col = 0; col < A_COLS; col++)
        {
            IDX_MAP[col] = IDX_MAP[col] - IDX_OFFSET[col];
        }
        {
            for (int j = row_block_coo_start_t.y; j < row_block_coo_end_t.y; ++j)
            {
                ecr_indices[j] = IDX_MAP[matA_csr->indices[j]];
            }
        }
        int p = 0;
        for (perfSpB_index col = 0; col < A_COLS; col++)
        {
            if (not_null_col_flag[col] != 1)
            {
                use_x_indices[row_blk_id][p] = col;
                p++;
            }
        }
        NEC_NUM[row_blk_id] = p;
        acc_nec_num += p;
        ecr_xx_val[row_blk_id] = (float *)malloc(p * sizeof(float));
        free(not_null_col_flag);
        free(IDX_MAP);
        free(IDX_OFFSET);
    }
    // 转换为一维
    int * row_blk_ptr =  (int *)malloc((NUM_ROW_BLOCKS) * sizeof(int));
    // int compact_x_dim = (NUM_ROW_BLOCKS * A_COLS + 1);// TODO: 太多冗余了
    int *h_use_x_indices = (int *)malloc((acc_nec_num) * sizeof(int));
    float *h_ecr_xx_val = (float *)malloc((acc_nec_num) * sizeof(float));
    memset(h_use_x_indices, 0, (acc_nec_num) * sizeof(int));
    memset(h_ecr_xx_val, 0.0, (acc_nec_num) * sizeof(float));
    perfSpB_index p = 0, acc = 0;
    for (int row_blk_id = 0; row_blk_id < NUM_ROW_BLOCKS; row_blk_id++)
    {
        int nec_num = NEC_NUM[row_blk_id];
        row_blk_ptr[row_blk_id] = acc;
        for (int j = 0; j < nec_num; j++)
        {
            h_use_x_indices[p++] = use_x_indices[row_blk_id][j];
        }
        acc += nec_num;
    }

    perfSpB_index *h_ROW_BLOCK_ID = (perfSpB_index *)malloc((A_ROWS) * sizeof(perfSpB_index));
    memset(h_ROW_BLOCK_ID, 0, (A_ROWS) * sizeof(perfSpB_index));

    for (int row_blk_id = 0; row_blk_id < NUM_ROW_BLOCKS; row_blk_id++)
    {
        for (int row = row_block_coo_start[row_blk_id].x; row < row_block_coo_end[row_blk_id].x; ++row)
        {
            h_ROW_BLOCK_ID[row] = row_blk_id;
        }
    }

    /////////////////////////////////////VNEC Kernel/////////////////////////////////////

    int *d_NEC_NUM;
    int *d_use_x_indices;
    int *d_row_blk_ptr;
    float *d_ecr_xx_val;
    float *d_vecX_csr;

    hipMalloc(&d_NEC_NUM, (NUM_ROW_BLOCKS + 1) * sizeof(int));
    hipMalloc(&d_use_x_indices, acc_nec_num * sizeof(int));
    hipMalloc(&d_row_blk_ptr, NUM_ROW_BLOCKS * sizeof(int));
    hipMalloc(&d_ecr_xx_val, acc_nec_num * sizeof(float));
    hipMalloc(&d_vecX_csr, sizeof(float) * vecX->n);
    hipMemcpy(d_NEC_NUM, NEC_NUM, sizeof(int) * (NUM_ROW_BLOCKS + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_use_x_indices, h_use_x_indices, sizeof(int) * acc_nec_num, hipMemcpyHostToDevice);
    hipMemcpy(d_row_blk_ptr, row_blk_ptr, sizeof(int) * NUM_ROW_BLOCKS, hipMemcpyHostToDevice);
    hipMemcpy(d_ecr_xx_val, h_ecr_xx_val, sizeof(float) * acc_nec_num, hipMemcpyHostToDevice);
    hipMemcpy(d_vecX_csr, ((float *)vecX_csr->values), sizeof(float) * vecX->n, hipMemcpyHostToDevice);
    



    float *d_vecY_csr, *d_val;
    perfSpB_index *d_ptr, *d_ROW_BLOCK_ID;
    int * d_ecr_indices;
    hipMalloc(&d_vecY_csr, sizeof(float) * vecY->n);
    hipMalloc(&d_val, sizeof(float) * matA_csr->nnz);
    hipMalloc(&d_ptr, sizeof(perfSpB_index) * (vecY->n + 1));
    hipMalloc(&d_ROW_BLOCK_ID, sizeof(perfSpB_index) * A_ROWS);
    hipMalloc(&d_ecr_indices, (nvals + 10) * sizeof(int));
    hipMemcpy(d_val, matA_csr->val, sizeof(float) * matA_csr->nnz, hipMemcpyHostToDevice);
    hipMemcpy(d_ptr, matA_csr->ptr, sizeof(perfSpB_index) * (vecY->n + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_ROW_BLOCK_ID, h_ROW_BLOCK_ID, sizeof(perfSpB_index) * A_ROWS, hipMemcpyHostToDevice);
    hipMemcpy(d_ecr_indices, ecr_indices, sizeof(int) * (nvals + 10), hipMemcpyHostToDevice);

    int mean_col_num = (matA_csr->nnz + (vecY->n - 1)) / vecY->n;
    printf("\n  The average col num is: %d\n", mean_col_num);
    int block_size_ = 256;
    unsigned int NUM_BLOCKS = static_cast<unsigned int>((acc_nec_num + (block_size_ - 1)) / block_size_);
    // vector_preprocess_kernel<float><<<NUM_BLOCKS, 256>>>(d_ecr_xx_val, d_use_x_indices, d_NEC_NUM, d_vecX_csr, d_row_blk_ptr);
    cuda_time_test_start();

    for (int i = 0; i < test_iter; i++)
    {
        // hipMemset(d_vecY_csr, 0.0, sizeof(float) * (vecY->n)); //TODO: Accuracy test
        vector_preprocess_kernel<float><<<NUM_BLOCKS, block_size_>>>(d_ecr_xx_val, d_use_x_indices, d_vecX_csr, acc_nec_num);
        
        if (mean_col_num <= 2)
        {
            const int THREADS_PER_VECTOR = 2;
            const unsigned int VECTORS_PER_BLOCK = 128;
            const unsigned int NUM_BLOCKS = static_cast<unsigned int>((vecY->n + (VECTORS_PER_BLOCK - 1)) / VECTORS_PER_BLOCK);
            vnec_spmv<float, VECTORS_PER_BLOCK, THREADS_PER_VECTOR><<<NUM_BLOCKS, 256>>>(d_vecY_csr, d_val, d_ptr, d_ecr_indices, d_ecr_xx_val, vecY->n, d_ROW_BLOCK_ID, d_row_blk_ptr);
        }
        else if (mean_col_num > 2 && mean_col_num <= 4)
        {
            const int THREADS_PER_VECTOR = 4;
            const unsigned int VECTORS_PER_BLOCK = 64;
            const unsigned int NUM_BLOCKS = static_cast<unsigned int>((vecY->n + (VECTORS_PER_BLOCK - 1)) / VECTORS_PER_BLOCK);
            vnec_spmv<float, VECTORS_PER_BLOCK, THREADS_PER_VECTOR><<<NUM_BLOCKS, 256>>>(d_vecY_csr, d_val, d_ptr, d_ecr_indices, d_ecr_xx_val, vecY->n, d_ROW_BLOCK_ID, d_row_blk_ptr);
        }
        else if (mean_col_num > 4 && mean_col_num <= 8)
        {
            const int THREADS_PER_VECTOR = 8;
            const unsigned int VECTORS_PER_BLOCK = 32;
            const unsigned int NUM_BLOCKS = static_cast<unsigned int>((vecY->n + (VECTORS_PER_BLOCK - 1)) / VECTORS_PER_BLOCK);
            vnec_spmv<float, VECTORS_PER_BLOCK, THREADS_PER_VECTOR><<<NUM_BLOCKS, 256>>>(d_vecY_csr, d_val, d_ptr, d_ecr_indices, d_ecr_xx_val, vecY->n, d_ROW_BLOCK_ID, d_row_blk_ptr);
        }
        else if (mean_col_num > 8 && mean_col_num <= 16)
        {
            const int THREADS_PER_VECTOR = 16;
            const unsigned int VECTORS_PER_BLOCK = 16;
            const unsigned int NUM_BLOCKS = static_cast<unsigned int>((vecY->n + (VECTORS_PER_BLOCK - 1)) / VECTORS_PER_BLOCK);
            vnec_spmv<float, VECTORS_PER_BLOCK, THREADS_PER_VECTOR><<<NUM_BLOCKS, 256>>>(d_vecY_csr, d_val, d_ptr, d_ecr_indices, d_ecr_xx_val, vecY->n, d_ROW_BLOCK_ID, d_row_blk_ptr);
        }
        else if (mean_col_num > 16 && mean_col_num <= 64)
        {
            const int THREADS_PER_VECTOR = 32;
            const unsigned int VECTORS_PER_BLOCK = 8;
            const unsigned int NUM_BLOCKS = static_cast<unsigned int>((vecY->n + (VECTORS_PER_BLOCK - 1)) / VECTORS_PER_BLOCK);
            vnec_spmv<float, VECTORS_PER_BLOCK, THREADS_PER_VECTOR><<<NUM_BLOCKS, 256>>>(d_vecY_csr, d_val, d_ptr, d_ecr_indices, d_ecr_xx_val, vecY->n, d_ROW_BLOCK_ID, d_row_blk_ptr);
        }
        else if (mean_col_num > 64)
    
        {
            const int THREADS_PER_VECTOR = 256;
            const unsigned int VECTORS_PER_BLOCK = 1;
            const unsigned int NUM_BLOCKS = static_cast<unsigned int>((vecY->n + (VECTORS_PER_BLOCK - 1)) / VECTORS_PER_BLOCK);
            vnec_spmv_L<float, VECTORS_PER_BLOCK, THREADS_PER_VECTOR><<<NUM_BLOCKS, 256>>>(d_vecY_csr, d_val, d_ptr, d_ecr_indices, d_ecr_xx_val, vecY->n, d_ROW_BLOCK_ID, d_row_blk_ptr);
        }
    }
    cuda_time_test_end();
    
    // hipMemcpy(h_ecr_xx_val, d_ecr_xx_val, sizeof(float) * acc_nec_num, hipMemcpyDeviceToHost);
    // for (int row_blk_id = 0; row_blk_id < NUM_ROW_BLOCKS; row_blk_id++)
    // {

    //     perfSpB_index row_blk_start = row_blk_ptr[row_blk_id];
    //     float *local_ecr_xx_val = h_ecr_xx_val + row_blk_start;
    //     float t1, t2, t3;
    //     for (int i = row_block_coo_start[row_blk_id].x; i < row_block_coo_end[row_blk_id].x; ++i)
    //     {
    //         t1 = 0.0f;
    //         perfSpB_index ptr_start = matA_csr->ptr[i];
    //         perfSpB_index n_one_line = matA_csr->ptr[i + 1] - ptr_start;

    //         for (perfSpB_index j = 0; j < n_one_line; j++)
    //         {
    //             t2 = ((float *)matA_csr->val)[j + ptr_start];
    //             t3 = 0.0f;
    //             perfSpB_index v_idx = ecr_indices[j + ptr_start];
    //             t3 = local_ecr_xx_val[v_idx];
    //             t1 += t2 * t3;
    //         }
    //         ((float *)vecY_csr->values)[i] = t1;
    //     }
    // }


    /////////////////////////////////////CUDA 计算结束/////////////////////////////////////
    double runtime = (elapsedTime) / test_iter;
    double gflops = (2.0 * matA_csr->nnz) / ((runtime / 1000) * 1e9);
    hipDeviceSynchronize();
    printf("\n -------------------------------------------------- \n");
    printf(" SpMV VNEC CUDA kernel runtime = %g ms\n", runtime);
    printf("\n SpMV VNEC Performance  = %lf GFLOPS", gflops);
    printf("\n -------------------------------------------------- \n");


    hipMemcpy(((float *)vecY_csr->values), d_vecY_csr, sizeof(float) * vecY->n, hipMemcpyDeviceToHost);

    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        printf("Error:%s\n", hipGetErrorString(cudaStatus));
        exit(EXIT_FAILURE);
    }
    hipFree(d_vecY_csr);

    hipFree(d_NEC_NUM);
    hipFree(d_use_x_indices);
    hipFree(d_ecr_xx_val);
    hipFree(d_vecX_csr);

    hipFree(d_val);
    hipFree(d_ecr_indices);
    hipFree(d_ptr);
    hipFree(d_ROW_BLOCK_ID);


    

    return perfSpB_success;
}