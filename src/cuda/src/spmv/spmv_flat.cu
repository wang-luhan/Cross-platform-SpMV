#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <spmv_cuda.h>
#include <math.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <algorithm>  // 包含 std::sort
#include <functional> // 包含 std::greater

#include "cuda_timer.h"

#define test_iter 2000

template <int BREAK_STRIDE, typename I>
__global__ void pre_startRowPerBlock(const I *__restrict__ row_ptr, const I m, I *__restrict__ startRowPerBlock)
{
  const int global_thread_id = threadIdx.x + blockDim.x * blockIdx.x;
  const int global_threads_num = blockDim.x * gridDim.x;

  constexpr I break_stride = BREAK_STRIDE;
  if (global_thread_id == 0)
  {
    startRowPerBlock[0] = 0; // start row of the block 0 and the first round.
  }

  for (int i = global_thread_id; i < m; i += global_threads_num) // 524,288
  {
    // for first element of row i and row i+1, they belong to different blocks.
    if (row_ptr[i] / break_stride != row_ptr[i + 1] / break_stride)
    { // fixme: step may be not 1
      // record the row id of the first element in the block.
      // note: a row can cross multiple blocks.
      for (int b = row_ptr[i] / break_stride + 1; b <= row_ptr[i + 1] / break_stride; b++)
      {
        startRowPerBlock[b] = i;
      }
      if (row_ptr[i + 1] % break_stride == 0)
      {
        startRowPerBlock[row_ptr[i + 1] / break_stride] += 1;
      }
    }
  }
}

// template <int BREAK_STRIDE, typename I>
// __global__ void pre_startRowPerBlock(const I *__restrict__ row_ptr,
//                                      const I m,
//                                      I *__restrict__ startRowPerBlock)
// {
//   const int global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
//   if (global_thread_id > m + 1)
//     return;
//   int a = row_ptr[global_thread_id];
//   int b = row_ptr[min(global_thread_id + 1, (int)m + 1)];

//   int blocka = divup<int>(a, BREAK_STRIDE);
//   int blockb = (b - 1) / static_cast<int>(BREAK_STRIDE);

//   if (a != b)
//     for (; blocka <= blockb; ++blocka)
//       startRowPerBlock[blocka] = global_thread_id;
// }

template <typename I, typename T, perfSpB_index NNZ_PER_BLOCK, perfSpB_index THREADS_PER_BLOCK>
__device__ __forceinline__ void flat_reduce_oneRow_in_thread(const int tid_in_block, const int block_id,
                                                             const I reduceStartRowId, const I reduceEndRowId,
                                                             const I *__restrict__ row_ptr,
                                                             const T *__restrict__ smem, T *__restrict__ y)
{
  I reduce_row_id = reduceStartRowId + tid_in_block;
  I nnz_id_before = block_id * NNZ_PER_BLOCK;
  for (; reduce_row_id < reduceEndRowId; reduce_row_id += THREADS_PER_BLOCK)
  {
    T sum = 0;
    // const I reduce_start_idx = max((perfSpB_index)0, row_ptr[reduce_row_id] - nnz_id_before);
    // const I reduce_end_idx = min(NNZ_PER_BLOCK, row_ptr[reduce_row_id + 1] - nnz_id_before);
    const I reduce_start_idx = (row_ptr[reduce_row_id] - nnz_id_before) < 0 ? 0 : (row_ptr[reduce_row_id] - nnz_id_before);
    const I reduce_end_idx = (row_ptr[reduce_row_id + 1] - nnz_id_before) > NNZ_PER_BLOCK ? NNZ_PER_BLOCK : (row_ptr[reduce_row_id + 1] - nnz_id_before);
    for (int i = reduce_start_idx; i < reduce_end_idx; i++)
    {
      sum += smem[i];
    }
    atomicAdd(y + reduce_row_id, sum);
  }
}
template <perfSpB_index productNnzPerThread, perfSpB_index THREADS_PER_BLOCK, typename I, typename T>
__global__ void spmv_flat_kernel(T *d_val,
                                 perfSpB_index *d_ptr,
                                 perfSpB_index *d_cols,
                                 perfSpB_index nrows,
                                 T *d_vector,
                                 T *d_out,
                                 I *__restrict__ startRowPerBlock)
{
  const int tid_in_block = threadIdx.x;
  const int NNZ_PER_BLOCK = THREADS_PER_BLOCK * productNnzPerThread;
  __shared__ T middle_s[NNZ_PER_BLOCK];
  const I lastElemId = d_ptr[nrows];

  int blockNnzStart = NNZ_PER_BLOCK * blockIdx.x;

  // product and stream in Shared Memory
#pragma unroll
  for (int round = 0; round < productNnzPerThread; round++)
  {
    const I sIdx = tid_in_block + round * THREADS_PER_BLOCK;
    const I gIdx = min(blockNnzStart + sIdx, lastElemId - 1);
    middle_s[sIdx] = d_val[gIdx] * d_vector[d_cols[gIdx]];
  }
  __syncthreads();

  const I reduceStartRowId = min(startRowPerBlock[blockIdx.x], nrows);
  I reduceEndRowId = min(startRowPerBlock[blockIdx.x + 1], nrows);
  reduceEndRowId = (reduceEndRowId == 0) ? nrows : reduceEndRowId;
  if (d_ptr[reduceEndRowId] % NNZ_PER_BLOCK != 0 || reduceEndRowId == reduceStartRowId)
  {
    reduceEndRowId = min(reduceEndRowId + 1, nrows);
  }

  flat_reduce_oneRow_in_thread<I, T, NNZ_PER_BLOCK, THREADS_PER_BLOCK>(tid_in_block, blockIdx.x,
                                                                       reduceStartRowId, reduceEndRowId,
                                                                       d_ptr, middle_s, d_out);
}

perfSpB_info perfSpB_SpMV_flat_FP32(perfSpB_operation_t op,
                                    const void *alpha,
                                    const perfSpB_matrix matA,
                                    const perfSpB_vector vecX,
                                    const void *beta,
                                    perfSpB_vector vecY)
{
  UNUSED(op);
  UNUSED(alpha);
  UNUSED(beta);
  perfSpB_vector_Dense *vecY_csr = (perfSpB_vector_Dense *)(vecY);
  perfSpB_vector_Dense *vecX_csr = (perfSpB_vector_Dense *)(vecX);
  perfSpB_matrix_CSC_or_CSR *matA_csr = (perfSpB_matrix_CSC_or_CSR *)(matA);
  perfSpB_index nrows = matA->row;
  // perfSpB_index ncols = matA->col;
  perfSpB_index nvals = matA_csr->nnz;

  float *d_vecY_csr, *d_vecX_csr, *d_val;
  perfSpB_index *d_indices, *d_ptr;
  for (int i = 0; i < matA->row; i++)
  {
    if (*((float *)vecY_csr->values) != 0)
    {
      printf("\n %f \n", *((float *)vecY_csr->values));
    }
  }

  hipMalloc(&d_vecY_csr, sizeof(float) * vecY->n);
  hipMalloc(&d_vecX_csr, sizeof(float) * vecX->n);
  hipMalloc(&d_val, sizeof(float) * matA_csr->nnz);
  hipMalloc(&d_indices, sizeof(perfSpB_index) * matA_csr->nnz);
  hipMalloc(&d_ptr, sizeof(perfSpB_index) * (vecY->n + 1));

  hipMemcpy(d_val, matA_csr->val, sizeof(float) * matA_csr->nnz, hipMemcpyHostToDevice);
  hipMemcpy(d_indices, matA_csr->indices, sizeof(perfSpB_index) * matA_csr->nnz, hipMemcpyHostToDevice);
  hipMemcpy(d_ptr, matA_csr->ptr, sizeof(perfSpB_index) * (vecY->n + 1), hipMemcpyHostToDevice);
  hipMemcpy(d_vecX_csr, ((float *)vecX_csr->values), sizeof(float) * vecX->n, hipMemcpyHostToDevice);
  hipMemcpy(d_vecY_csr, ((float *)vecY_csr->values), sizeof(float) * vecY->n, hipMemcpyHostToDevice);

  const int productNnzPerThread = 4;
  const int THREADS_PER_BLOCK = 128;

  const int WORK_BLOCKS = nvals / (productNnzPerThread * THREADS_PER_BLOCK) + ((nvals % (productNnzPerThread * THREADS_PER_BLOCK) == 0) ? 0 : 1);

  const perfSpB_index startRowPerBlock_len = WORK_BLOCKS + 1;

  perfSpB_index *startRowPerBlock;
  hipMalloc((void **)&startRowPerBlock, sizeof(perfSpB_index) * startRowPerBlock_len);
  hipMemset(startRowPerBlock, 0, sizeof(perfSpB_index) * startRowPerBlock_len);

  pre_startRowPerBlock<productNnzPerThread * THREADS_PER_BLOCK, perfSpB_index><<<divup<uint32_t>(nrows + 1, 256), 256>>>(d_ptr, nrows, startRowPerBlock);

  // perfSpB_index *h_startRowPerBlock = (perfSpB_index *)malloc((startRowPerBlock_len) * sizeof(perfSpB_index));
  // hipMemcpy(h_startRowPerBlock, startRowPerBlock, (startRowPerBlock_len) * sizeof(perfSpB_index), hipMemcpyDeviceToHost);

  /*
  /////////////////////////////////////////////////////////////////////////////////////////////////
  // 打印超长行
  int *row_len = (int *)malloc((nrows) * sizeof(int));
  std::vector<std::pair<int, int>> pairs(nrows);
  for (int i = 0; i < nrows; i++)
  {
    row_len[i] = matA_csr->ptr[i + 1] - matA_csr->ptr[i];
    pairs[i] = std::make_pair(row_len[i], i);
  }
  std::sort(pairs.begin(), pairs.end(), std::greater<std::pair<int, int>>());

  int count = 0;
  for (const auto &pair : pairs)
  {
    printf("row %d has %d nnz\n", pair.second, pair.first);
    count++;
    if (count == 10)
      break;
  }
  double mean_col_num = ((double)matA_csr->nnz + (double)(vecY->n - 1)) / (double)vecY->n;
  printf("\n  The average col num is: %lf\n", mean_col_num);
  /////////////////////////////////////////////////////////////////////////////////////////////////
  */
  cuda_time_test_start();
  for (int i = 0; i < test_iter; i++)
  {
    // hipMemset(d_vecY_csr, 0.0, sizeof(float) * vecY->n);
    spmv_flat_kernel<productNnzPerThread, THREADS_PER_BLOCK, perfSpB_index, float><<<(WORK_BLOCKS), (THREADS_PER_BLOCK)>>>(d_val, d_ptr, d_indices, nrows, d_vecX_csr, d_vecY_csr, startRowPerBlock);
  }
  cuda_time_test_end();
  double runtime = (elapsedTime) / test_iter;
  double gflops = (2.0 * matA_csr->nnz) / ((runtime / 1000) * 1e9);

  hipDeviceSynchronize();
  printf("\n SpMV CUDA kernel runtime = %g ms\n", runtime);
  // printf("\n SpMV CUDA kernel2 runtime = %g ms\n", runtime2);
  printf("\n SpMV Performance  = %lf GFLOPS\n", gflops);

  hipMemcpy(((float *)vecY_csr->values), d_vecY_csr, sizeof(float) * vecY->n, hipMemcpyDeviceToHost);

  hipError_t cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess)
  {
    printf("Error:%s\n", hipGetErrorString(cudaStatus));
    exit(EXIT_FAILURE);
  }

  hipFree(d_vecY_csr);
  hipFree(d_vecX_csr);
  hipFree(d_val);
  hipFree(d_indices);
  hipFree(d_ptr);
  hipFree(startRowPerBlock);

  return perfSpB_success;
}

perfSpB_info perfSpB_SpMV_flat_FP64(perfSpB_operation_t op,
                                    const void *alpha,
                                    const perfSpB_matrix matA,
                                    const perfSpB_vector vecX,
                                    const void *beta,
                                    perfSpB_vector vecY)
{
  UNUSED(op);
  UNUSED(alpha);
  UNUSED(beta);
  perfSpB_vector_Dense *vecY_csr = (perfSpB_vector_Dense *)(vecY);
  perfSpB_vector_Dense *vecX_csr = (perfSpB_vector_Dense *)(vecX);
  perfSpB_matrix_CSC_or_CSR *matA_csr = (perfSpB_matrix_CSC_or_CSR *)(matA);
  perfSpB_index nrows = matA->row;
  // perfSpB_index ncols = matA->col;
  perfSpB_index nvals = matA_csr->nnz;

  double *d_vecY_csr, *d_vecX_csr, *d_val;
  perfSpB_index *d_indices, *d_ptr;
  for (int i = 0; i < matA->row; i++)
  {
    if (*((double *)vecY_csr->values) != 0)
    {
      printf("\n %f \n", *((double *)vecY_csr->values));
    }
  }

  hipMalloc(&d_vecY_csr, sizeof(double) * vecY->n);
  hipMalloc(&d_vecX_csr, sizeof(double) * vecX->n);
  hipMalloc(&d_val, sizeof(double) * matA_csr->nnz);
  hipMalloc(&d_indices, sizeof(perfSpB_index) * matA_csr->nnz);
  hipMalloc(&d_ptr, sizeof(perfSpB_index) * (vecY->n + 1));

  hipMemcpy(d_val, matA_csr->val, sizeof(double) * matA_csr->nnz, hipMemcpyHostToDevice);
  hipMemcpy(d_indices, matA_csr->indices, sizeof(perfSpB_index) * matA_csr->nnz, hipMemcpyHostToDevice);
  hipMemcpy(d_ptr, matA_csr->ptr, sizeof(perfSpB_index) * (vecY->n + 1), hipMemcpyHostToDevice);
  hipMemcpy(d_vecX_csr, ((double *)vecX_csr->values), sizeof(double) * vecX->n, hipMemcpyHostToDevice);
  hipMemcpy(d_vecY_csr, ((double *)vecY_csr->values), sizeof(double) * vecY->n, hipMemcpyHostToDevice);

  const int productNnzPerThread = 4;
  const int THREADS_PER_BLOCK = 128;

  const int WORK_BLOCKS = nvals / (productNnzPerThread * THREADS_PER_BLOCK) + ((nvals % (productNnzPerThread * THREADS_PER_BLOCK) == 0) ? 0 : 1);

  const perfSpB_index startRowPerBlock_len = WORK_BLOCKS + 1;

  perfSpB_index *startRowPerBlock;
  hipMalloc((void **)&startRowPerBlock, sizeof(perfSpB_index) * startRowPerBlock_len);
  hipMemset(startRowPerBlock, 0, sizeof(perfSpB_index) * startRowPerBlock_len);

  pre_startRowPerBlock<productNnzPerThread * THREADS_PER_BLOCK, perfSpB_index><<<divup<uint32_t>(nrows + 1, 256), 256>>>(d_ptr, nrows, startRowPerBlock);


  cuda_time_test_start();
  for (int i = 0; i < test_iter; i++)
  {
    // hipMemset(d_vecY_csr, 0.0, sizeof(double) * vecY->n);
    spmv_flat_kernel<productNnzPerThread, THREADS_PER_BLOCK, perfSpB_index, double><<<(WORK_BLOCKS), (THREADS_PER_BLOCK)>>>(d_val, d_ptr, d_indices, nrows, d_vecX_csr, d_vecY_csr, startRowPerBlock);
  }
  cuda_time_test_end();
  double runtime = (elapsedTime) / test_iter;
  double gflops = (2.0 * matA_csr->nnz) / ((runtime / 1000) * 1e9);

  hipDeviceSynchronize();
  printf("\n SpMV CUDA kernel runtime = %g ms\n", runtime);
  // printf("\n SpMV CUDA kernel2 runtime = %g ms\n", runtime2);
  printf("\n SpMV Performance  = %lf GFLOPS\n", gflops);

  hipMemcpy(((double *)vecY_csr->values), d_vecY_csr, sizeof(double) * vecY->n, hipMemcpyDeviceToHost);

  hipError_t cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess)
  {
    printf("Error:%s\n", hipGetErrorString(cudaStatus));
    exit(EXIT_FAILURE);
  }

  hipFree(d_vecY_csr);
  hipFree(d_vecX_csr);
  hipFree(d_val);
  hipFree(d_indices);
  hipFree(d_ptr);
  hipFree(startRowPerBlock);

  return perfSpB_success;
}
