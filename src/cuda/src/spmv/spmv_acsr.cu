#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <spmv_cuda.h>
#include "cuda_timer.h"

#define test_iter 1
#define BlockDim 1024
template <unsigned int threads_per_row>
__device__ __forceinline__ float warpReduceSum(float sum)
{
    if (threads_per_row >= 32)
        sum += __shfl_down_sync(0xffffffff, sum, 16); // 0-16, 1-17, 2-18, etc.
    if (threads_per_row >= 16)
        sum += __shfl_down_sync(0xffffffff, sum, 8); // 0-8, 1-9, 2-10, etc.
    if (threads_per_row >= 8)
        sum += __shfl_down_sync(0xffffffff, sum, 4); // 0-4, 1-5, 2-6, etc.
    if (threads_per_row >= 4)
        sum += __shfl_down_sync(0xffffffff, sum, 2); // 0-2, 1-3, 4-6, 5-7, etc.
    if (threads_per_row >= 2)
        sum += __shfl_down_sync(0xffffffff, sum, 1); // 0-1, 2-3, 4-5, etc.
    return sum;
}

template <typename ValueType, unsigned int VECTORS_PER_BLOCK, unsigned int THREADS_PER_VECTOR>
__global__ void spmv_kernel_small(ValueType *__restrict__ y,
                                  const ValueType *__restrict__ A_value,
                                  const perfSpB_index *__restrict__ A_col_index,
                                  const perfSpB_index *__restrict__ A_row_offset,
                                  perfSpB_index *A_row_id,
                                  const ValueType *__restrict__ x,
                                  perfSpB_index row_num)
{
    const perfSpB_index thread_id = blockDim.x * blockIdx.x + threadIdx.x;    // global thread index
    const perfSpB_index thread_lane = threadIdx.x & (THREADS_PER_VECTOR - 1); // thread index within the vector/row
    const perfSpB_index row_id = thread_id / THREADS_PER_VECTOR;              // global vector index
    if (row_id < row_num && row_id != 0)
    {
        perfSpB_index real_row_id = A_row_id[row_id];
        const perfSpB_index row_start = A_row_offset[real_row_id]; // same as: row_start = Ap[row];
        const perfSpB_index row_end = A_row_offset[real_row_id + 1];

        // initialize local sum
        ValueType sum = 0;

        // accumulate local sums
        for (perfSpB_index jj = row_start + thread_lane; jj < row_end; jj += THREADS_PER_VECTOR)
            sum += A_value[jj] * x[A_col_index[jj]];

        sum = warpReduceSum<THREADS_PER_VECTOR>(sum);
        if (thread_lane == 0)
        {
            y[real_row_id] = sum;
        }
    }
}
template <typename T>
__global__ __forceinline__ void spmv_kernel_large(T *__restrict__ y,
                                                  const T *__restrict__ A_value,
                                                  const perfSpB_index *__restrict__ A_col_index,
                                                  const perfSpB_index *__restrict__ A_row_offset,
                                                  perfSpB_index *A_row_id,
                                                  const T *__restrict__ x,
                                                  perfSpB_index row_num)
{
    int real_row_id = A_row_id[blockIdx.x];
    __shared__ volatile T LDS[BlockDim];
    int rowStart = A_row_offset[real_row_id];
    int rowEnd = A_row_offset[real_row_id + 1];

    T sum = 0;
    for (int j = rowStart + threadIdx.x; j < rowEnd; j += BlockDim)
    {
        int col = A_col_index[j];
        sum += A_value[j] * x[col];
    }

    LDS[threadIdx.x] = sum;
    __syncthreads();

    // Reduce partial sums
    for (int stride = blockDim.x >> 1; stride > 0; stride >>= 1)
    {
        __syncthreads();
        if (threadIdx.x < stride)
            LDS[threadIdx.x] += LDS[threadIdx.x + stride];
    }
    // Write result
    if (threadIdx.x == 0)
        atomicAdd(y + real_row_id, LDS[threadIdx.x]);
}

// template <typename ValueType, unsigned int VECTORS_PER_BLOCK, unsigned int THREADS_PER_VECTOR>
// __global__ void spmv_kernel_cuda_L(ValueType *__restrict__ y,
//                                    const ValueType *__restrict__ A_value,
//                                    const perfSpB_index *__restrict__ A_col_index,
//                                    const perfSpB_index *__restrict__ A_row_offset,
//                                    const ValueType *__restrict__ x,
//                                    perfSpB_index row_num)
// {
//     const perfSpB_index thread_id = blockDim.x * blockIdx.x + threadIdx.x;    // global thread index
//     const perfSpB_index thread_lane = threadIdx.x & (THREADS_PER_VECTOR - 1); // thread index within the vector/row
//     const perfSpB_index row_id = thread_id / THREADS_PER_VECTOR;              // global vector index
//     const perfSpB_index lane_id = threadIdx.x % 32;                           // global vector index

//     if (row_id < row_num)
//     {
//         const perfSpB_index row_start = A_row_offset[row_id]; // same as: row_start = Ap[row];
//         const perfSpB_index row_end = A_row_offset[row_id + 1];

//         // initialize local sum
//         ValueType sum = 0;

//         // accumulate local sums
//         for (perfSpB_index jj = row_start + thread_lane; jj < row_end; jj += THREADS_PER_VECTOR)
//             sum += A_value[jj] * x[A_col_index[jj]];

//         for (int offset = 16; offset > 0; offset >>= 1)
//         {
//             sum += __shfl_down_sync(0xffffffff, sum, offset);
//         }

//         if (lane_id == 0)
//         {
//             atomicAdd(&y[row_id], sum);
//             // y[row_id] = sum;
//             // y[row_id] = s_tmp[0] + s_tmp[1] + s_tmp[2] + s_tmp[3];
//         }
//     }
// }

perfSpB_info perfSpB_SpMV_ACSR_FP32(perfSpB_operation_t op,
                                    const void *alpha,
                                    const perfSpB_matrix matA,
                                    const perfSpB_vector vecX,
                                    const void *beta,
                                    perfSpB_vector vecY)
{
    UNUSED(op);
    UNUSED(alpha);
    UNUSED(beta);

    perfSpB_vector_Dense *vecY_csr = (perfSpB_vector_Dense *)(vecY);
    perfSpB_vector_Dense *vecX_csr = (perfSpB_vector_Dense *)(vecX);
    perfSpB_matrix_CSC_or_CSR *matA_csr = (perfSpB_matrix_CSC_or_CSR *)(matA);
    float *d_vecY_csr, *d_vecX_csr, *d_val;
    perfSpB_index *d_indices, *d_ptr;
    for (int i = 0; i < matA->row; i++)
    {
        if (*((float *)vecY_csr->values) != 0)
        {
            printf("\n %f \n", *((float *)vecY_csr->values));
        }
    }

    hipMalloc(&d_vecY_csr, sizeof(float) * vecY->n);
    hipMalloc(&d_vecX_csr, sizeof(float) * vecX->n);
    hipMalloc(&d_val, sizeof(float) * matA_csr->nnz);
    hipMalloc(&d_indices, sizeof(perfSpB_index) * matA_csr->nnz);
    hipMalloc(&d_ptr, sizeof(perfSpB_index) * (vecY->n + 1));

    hipMemcpy(d_val, matA_csr->val, sizeof(float) * matA_csr->nnz, hipMemcpyHostToDevice);
    hipMemcpy(d_indices, matA_csr->indices, sizeof(perfSpB_index) * matA_csr->nnz, hipMemcpyHostToDevice);
    hipMemcpy(d_ptr, matA_csr->ptr, sizeof(perfSpB_index) * (vecY->n + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_vecX_csr, ((float *)vecX_csr->values), sizeof(float) * vecX->n, hipMemcpyHostToDevice);
    // hipMemcpy(d_vecY_csr, ((float *)vecY_csr->values), sizeof(float) * vecY->n, hipMemcpyHostToDevice);
    // hipMemset(d_vecY_csr, 0.0, sizeof(float) * (vecY->n));

    // spmv_kernel_cuda<<<blocks, threads>>>(d_vecY_csr, d_val, d_indices, d_ptr, d_vecX_csr, vecY->n, matA_csr->nnz);
    double mean_col_num = ((double)matA_csr->nnz + (double)(vecY->n - 1)) / (double)vecY->n;
    int mean_nnz_thresh = mean_col_num * 200;
    
    perfSpB_index *rowId_L = (perfSpB_index *)malloc(matA->row * sizeof(perfSpB_index));
    perfSpB_index *rowId_S = (perfSpB_index *)malloc(matA->row * sizeof(perfSpB_index));
    memset(rowId_L, 0, matA->row * sizeof(perfSpB_index));
    memset(rowId_S, 0, matA->row * sizeof(perfSpB_index));
    int p = 0, q = 0, Lrow_num = 0, Srow_num = 0;
    for (int i = 0; i < matA->row; i++)
    {
        int row_nnz_tmp = matA_csr->ptr[i + 1] - matA_csr->ptr[i];
        if (row_nnz_tmp > mean_nnz_thresh)
        {
            rowId_L[p++] = i;
        }
        else
        {
            rowId_S[q++] = i;
        }
    }
    
    Lrow_num = p;
    Srow_num = q;
    printf("\n %d   %d \n", Lrow_num, Srow_num);
    perfSpB_index *d_rowId_L, *d_rowId_S;
    hipMalloc(&d_rowId_L, sizeof(perfSpB_index) * Lrow_num);
    hipMalloc(&d_rowId_S, sizeof(perfSpB_index) * Srow_num);
    hipMemcpy(d_rowId_L, rowId_L, sizeof(perfSpB_index) * Lrow_num, hipMemcpyHostToDevice);
    hipMemcpy(d_rowId_S, rowId_S, sizeof(perfSpB_index) * Srow_num, hipMemcpyHostToDevice);
    hipStream_t stream_1;
    hipStream_t stream_2;
    hipStreamCreate(&stream_1);
    hipStreamCreate(&stream_2);

    // double accum = 0.0, max = 0.0;
    // for (int row_id = 1; row_id < matA->row; row_id++)
    // {
    //     double row_nnz_tmp = (double)matA_csr->ptr[row_id + 1] - (double)matA_csr->ptr[row_id];
    //     max = (max < row_nnz_tmp) ? row_nnz_tmp : max;
    //     accum += abs(row_nnz_tmp - mean_col_num);
    // }
    // printf("\n  The average col num is: %lf\n", mean_col_num);
    // // // printf("\n  The variance is: %lf\n", (accum / matA->row));
    // printf("\n  The max nnz per row is: %lf\n", (max));
    // printf("\n  lat_time: %lf\n", (max / mean_col_num));
    
    cuda_time_test_start();
    for (int i = 0; i < test_iter; i++)
    {
        // hipMemset(d_vecY_csr, 0.0, sizeof(float) * (vecY->n)); //TODO: Accuracy test
        spmv_kernel_large<float><<<Lrow_num, BlockDim, 0, stream_1>>>(d_vecY_csr, d_val, d_indices, d_ptr, d_rowId_L, d_vecX_csr, Lrow_num);
        
        if (mean_col_num <= 2)
        {
            const int THREADS_PER_VECTOR = 2;
            const unsigned int VECTORS_PER_BLOCK = 128;
            const unsigned int NUM_BLOCKS = static_cast<unsigned int>((Srow_num + (VECTORS_PER_BLOCK - 1)) / VECTORS_PER_BLOCK);
            spmv_kernel_small<float, VECTORS_PER_BLOCK, THREADS_PER_VECTOR><<<NUM_BLOCKS, 256, 0, stream_2>>>(d_vecY_csr, d_val, d_indices, d_ptr, d_rowId_S, d_vecX_csr, Srow_num);
        }
        else if (mean_col_num > 2 && mean_col_num <= 4)
        {
            const int THREADS_PER_VECTOR = 4;
            const unsigned int VECTORS_PER_BLOCK = 64;
            const unsigned int NUM_BLOCKS = static_cast<unsigned int>((Srow_num + (VECTORS_PER_BLOCK - 1)) / VECTORS_PER_BLOCK);
            spmv_kernel_small<float, VECTORS_PER_BLOCK, THREADS_PER_VECTOR><<<NUM_BLOCKS, 256, 0, stream_2>>>(d_vecY_csr, d_val, d_indices, d_ptr, d_rowId_S, d_vecX_csr, Srow_num);
        }
        else if (mean_col_num > 4 && mean_col_num <= 8)
        {
            const int THREADS_PER_VECTOR = 8;
            const unsigned int VECTORS_PER_BLOCK = 32;
            const unsigned int NUM_BLOCKS = static_cast<unsigned int>((Srow_num + (VECTORS_PER_BLOCK - 1)) / VECTORS_PER_BLOCK);
            spmv_kernel_small<float, VECTORS_PER_BLOCK, THREADS_PER_VECTOR><<<NUM_BLOCKS, 256, 0, stream_2>>>(d_vecY_csr, d_val, d_indices, d_ptr, d_rowId_S, d_vecX_csr, Srow_num);
        }
        else if (mean_col_num > 8 && mean_col_num <= 16)
        {
            const int THREADS_PER_VECTOR = 16;
            const unsigned int VECTORS_PER_BLOCK = 16;
            const unsigned int NUM_BLOCKS = static_cast<unsigned int>((Srow_num + (VECTORS_PER_BLOCK - 1)) / VECTORS_PER_BLOCK);
            spmv_kernel_small<float, VECTORS_PER_BLOCK, THREADS_PER_VECTOR><<<NUM_BLOCKS, 256, 0, stream_2>>>(d_vecY_csr, d_val, d_indices, d_ptr, d_rowId_S, d_vecX_csr, Srow_num);
        }
        else if (mean_col_num > 16)
        {
            const int THREADS_PER_VECTOR = 32;
            const unsigned int VECTORS_PER_BLOCK = 8;
            const unsigned int NUM_BLOCKS = static_cast<unsigned int>((Srow_num + (VECTORS_PER_BLOCK - 1)) / VECTORS_PER_BLOCK);
            spmv_kernel_small<float, VECTORS_PER_BLOCK, THREADS_PER_VECTOR><<<NUM_BLOCKS, 256, 0, stream_2>>>(d_vecY_csr, d_val, d_indices, d_ptr, d_rowId_S, d_vecX_csr, Srow_num);
        }
        // else if (mean_col_num > 32 && mean_col_num <= 64)
        // {
        //     const int THREADS_PER_VECTOR = 64;
        //     const unsigned int VECTORS_PER_BLOCK = 4;
        //     const unsigned int NUM_BLOCKS = static_cast<unsigned int>((vecY->n + (VECTORS_PER_BLOCK - 1)) / VECTORS_PER_BLOCK);
        //     spmv_kernel_cuda_L<float, VECTORS_PER_BLOCK, THREADS_PER_VECTOR><<<NUM_BLOCKS, 256>>>(d_vecY_csr, d_val, d_indices, d_ptr, d_vecX_csr, vecY->n);
        // }
        // else if (mean_col_num > 64)
        // {
        //     const int THREADS_PER_VECTOR = 256;
        //     const unsigned int VECTORS_PER_BLOCK = 1;
        //     const unsigned int NUM_BLOCKS = static_cast<unsigned int>((vecY->n + (VECTORS_PER_BLOCK - 1)) / VECTORS_PER_BLOCK);
        //     spmv_kernel_cuda_L<float, VECTORS_PER_BLOCK, THREADS_PER_VECTOR><<<NUM_BLOCKS, 256>>>(d_vecY_csr, d_val, d_indices, d_ptr, d_vecX_csr, vecY->n);
        // }
    }

    cuda_time_test_end();
    double runtime = (elapsedTime) / test_iter;
    double gflops = (2.0 * matA_csr->nnz) / ((runtime / 1000) * 1e9);
    hipDeviceSynchronize();
    printf("\n SpMV CUDA kernel runtime = %g ms\n", runtime);
    printf("\n SpMV Performance  = %lf GFLOPS\n", gflops);

    hipMemcpy(((float *)vecY_csr->values), d_vecY_csr, sizeof(float) * vecY->n, hipMemcpyDeviceToHost);

    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        printf("Error:%s\n", hipGetErrorString(cudaStatus));
        exit(EXIT_FAILURE);
    }
    hipFree(d_vecY_csr);
    hipFree(d_vecX_csr);
    hipFree(d_val);
    hipFree(d_indices);
    hipFree(d_ptr);
    hipStreamDestroy(stream_1);
    hipStreamDestroy(stream_2);

    return perfSpB_success;
}