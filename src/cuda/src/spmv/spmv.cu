#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <spmv_cuda.h>
#include "cuda_timer.h"

#define test_iter 2000

template <unsigned int threads_per_row>
__device__ __forceinline__ float warpReduceSum(float sum)
{
    if (threads_per_row >= 32)
        sum += __shfl_down_sync(0xffffffff, sum, 16); // 0-16, 1-17, 2-18, etc.
    if (threads_per_row >= 16)
        sum += __shfl_down_sync(0xffffffff, sum, 8); // 0-8, 1-9, 2-10, etc.
    if (threads_per_row >= 8)
        sum += __shfl_down_sync(0xffffffff, sum, 4); // 0-4, 1-5, 2-6, etc.
    if (threads_per_row >= 4)
        sum += __shfl_down_sync(0xffffffff, sum, 2); // 0-2, 1-3, 4-6, 5-7, etc.
    if (threads_per_row >= 2)
        sum += __shfl_down_sync(0xffffffff, sum, 1); // 0-1, 2-3, 4-5, etc.
    return sum;
}

template <typename ValueType, unsigned int VECTORS_PER_BLOCK, unsigned int THREADS_PER_VECTOR>
__global__ void spmv_kernel_cuda(ValueType *__restrict__ y,
                                 const ValueType *__restrict__ A_value,
                                 const perfSpB_index *__restrict__ A_col_index,
                                 const perfSpB_index *__restrict__ A_row_offset,
                                 const ValueType *__restrict__ x,
                                 perfSpB_index row_num)
{
    const perfSpB_index thread_id = blockDim.x * blockIdx.x + threadIdx.x;    // global thread index
    const perfSpB_index thread_lane = threadIdx.x & (THREADS_PER_VECTOR - 1); // thread index within the vector/row
    const perfSpB_index row_id = thread_id / THREADS_PER_VECTOR;              // global vector index
    if (row_id < row_num)
    {
        const perfSpB_index row_start = A_row_offset[row_id]; // same as: row_start = Ap[row];
        const perfSpB_index row_end = A_row_offset[row_id + 1];

        // initialize local sum
        ValueType sum = 0;

        // accumulate local sums
        for (perfSpB_index jj = row_start + thread_lane; jj < row_end; jj += THREADS_PER_VECTOR)
            sum += A_value[jj] * x[A_col_index[jj]];

        sum = warpReduceSum<THREADS_PER_VECTOR>(sum);
        if (thread_lane == 0)
        {
            y[row_id] = sum;
        }
    }
}

template <typename ValueType, unsigned int VECTORS_PER_BLOCK, unsigned int THREADS_PER_VECTOR>
__global__ void spmv_kernel_cuda_L(ValueType *__restrict__ y,
                                   const ValueType *__restrict__ A_value,
                                   const perfSpB_index *__restrict__ A_col_index,
                                   const perfSpB_index *__restrict__ A_row_offset,
                                   const ValueType *__restrict__ x,
                                   perfSpB_index row_num)
{
    const perfSpB_index thread_id = blockDim.x * blockIdx.x + threadIdx.x;    // global thread index
    const perfSpB_index thread_lane = threadIdx.x & (THREADS_PER_VECTOR - 1); // thread index within the vector/row
    const perfSpB_index row_id = thread_id / THREADS_PER_VECTOR;              // global vector index
    const perfSpB_index lane_id = threadIdx.x % 32;                           // global vector index

    // if(row_id < row_num && threadIdx.x == 0)
    // {
    //     if (y[row_id] != 0) printf("\ny[%ld] = %f\n", row_id,y[row_id]);
    // }

    if (row_id < row_num)
    {
        const perfSpB_index row_start = A_row_offset[row_id]; // same as: row_start = Ap[row];
        const perfSpB_index row_end = A_row_offset[row_id + 1];

        // initialize local sum
        ValueType sum = 0;

        // accumulate local sums
        for (perfSpB_index jj = row_start + thread_lane; jj < row_end; jj += THREADS_PER_VECTOR)
            sum += A_value[jj] * x[A_col_index[jj]];

        for (int offset = 16; offset > 0; offset >>= 1)
        {
            sum += __shfl_down_sync(0xffffffff, sum, offset);
        }

        if (lane_id == 0)
        {
            atomicAdd(&y[row_id], sum);
            // y[row_id] = sum;
            // y[row_id] = s_tmp[0] + s_tmp[1] + s_tmp[2] + s_tmp[3];
        }
    }
}

perfSpB_info perfSpB_SpMV_FP32(perfSpB_operation_t op,
                               const void *alpha,
                               const perfSpB_matrix matA,
                               const perfSpB_vector vecX,
                               const void *beta,
                               perfSpB_vector vecY)
{
    UNUSED(op);
    UNUSED(alpha);
    UNUSED(beta);

    perfSpB_vector_Dense *vecY_csr = (perfSpB_vector_Dense *)(vecY);
    perfSpB_vector_Dense *vecX_csr = (perfSpB_vector_Dense *)(vecX);
    perfSpB_matrix_CSC_or_CSR *matA_csr = (perfSpB_matrix_CSC_or_CSR *)(matA);
    float *d_vecY_csr, *d_vecX_csr, *d_val;
    perfSpB_index *d_indices, *d_ptr;
    for (int i = 0; i < matA->row; i++)
    {
        if (*((float *)vecY_csr->values) != 0)
        {
            printf("\n %f \n", *((float *)vecY_csr->values));
        }
    }

    hipMalloc(&d_vecY_csr, sizeof(float) * vecY->n);
    hipMalloc(&d_vecX_csr, sizeof(float) * vecX->n);
    hipMalloc(&d_val, sizeof(float) * matA_csr->nnz);
    hipMalloc(&d_indices, sizeof(perfSpB_index) * matA_csr->nnz);
    hipMalloc(&d_ptr, sizeof(perfSpB_index) * (vecY->n + 1));

    hipMemcpy(d_val, matA_csr->val, sizeof(float) * matA_csr->nnz, hipMemcpyHostToDevice);
    hipMemcpy(d_indices, matA_csr->indices, sizeof(perfSpB_index) * matA_csr->nnz, hipMemcpyHostToDevice);
    hipMemcpy(d_ptr, matA_csr->ptr, sizeof(perfSpB_index) * (vecY->n + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_vecX_csr, ((float *)vecX_csr->values), sizeof(float) * vecX->n, hipMemcpyHostToDevice);
    // hipMemcpy(d_vecY_csr, ((float *)vecY_csr->values), sizeof(float) * vecY->n, hipMemcpyHostToDevice);
    // hipMemset(d_vecY_csr, 0.0, sizeof(float) * (vecY->n));

    // spmv_kernel_cuda<<<blocks, threads>>>(d_vecY_csr, d_val, d_indices, d_ptr, d_vecX_csr, vecY->n, matA_csr->nnz);
    double mean_col_num = ((double)matA_csr->nnz + (double)(vecY->n - 1)) / (double)vecY->n;
    double accum = 0.0, max = 0.0;
    for (int row_id = 1; row_id < matA->row; row_id++)
    {
        double row_nnz_tmp = (double)matA_csr->ptr[row_id + 1] - (double)matA_csr->ptr[row_id];
        max = (max < row_nnz_tmp) ? row_nnz_tmp : max;
        accum += abs(row_nnz_tmp - mean_col_num);
    }
    // printf("\n  The average col num is: %lf\n", mean_col_num);
    // // printf("\n  The variance is: %lf\n", (accum / matA->row));
    // printf("\n  The max nnz per row is: %lf\n", (max));
    // printf("\n  lat_time: %lf\n", (max / mean_col_num));
    cuda_time_test_start();
    for (int i = 0; i < test_iter; i++)
    {
        // hipMemset(d_vecY_csr, 0.0, sizeof(float) * (vecY->n)); //TODO: Accuracy test

        if (mean_col_num <= 2)
        {
            const int THREADS_PER_VECTOR = 2;
            const unsigned int VECTORS_PER_BLOCK = 128;
            const unsigned int NUM_BLOCKS = static_cast<unsigned int>((vecY->n + (VECTORS_PER_BLOCK - 1)) / VECTORS_PER_BLOCK);
            spmv_kernel_cuda<float, VECTORS_PER_BLOCK, THREADS_PER_VECTOR><<<NUM_BLOCKS, 256>>>(d_vecY_csr, d_val, d_indices, d_ptr, d_vecX_csr, vecY->n);
        }
        else if (mean_col_num > 2 && mean_col_num <= 4)
        {
            const int THREADS_PER_VECTOR = 4;
            const unsigned int VECTORS_PER_BLOCK = 64;
            const unsigned int NUM_BLOCKS = static_cast<unsigned int>((vecY->n + (VECTORS_PER_BLOCK - 1)) / VECTORS_PER_BLOCK);
            spmv_kernel_cuda<float, VECTORS_PER_BLOCK, THREADS_PER_VECTOR><<<NUM_BLOCKS, 256>>>(d_vecY_csr, d_val, d_indices, d_ptr, d_vecX_csr, vecY->n);
        }
        else if (mean_col_num > 4 && mean_col_num <= 8)
        {
            const int THREADS_PER_VECTOR = 8;
            const unsigned int VECTORS_PER_BLOCK = 32;
            const unsigned int NUM_BLOCKS = static_cast<unsigned int>((vecY->n + (VECTORS_PER_BLOCK - 1)) / VECTORS_PER_BLOCK);
            spmv_kernel_cuda<float, VECTORS_PER_BLOCK, THREADS_PER_VECTOR><<<NUM_BLOCKS, 256>>>(d_vecY_csr, d_val, d_indices, d_ptr, d_vecX_csr, vecY->n);
        }
        else if (mean_col_num > 8 && mean_col_num <= 16)
        {
            const int THREADS_PER_VECTOR = 16;
            const unsigned int VECTORS_PER_BLOCK = 16;
            const unsigned int NUM_BLOCKS = static_cast<unsigned int>((vecY->n + (VECTORS_PER_BLOCK - 1)) / VECTORS_PER_BLOCK);
            spmv_kernel_cuda<float, VECTORS_PER_BLOCK, THREADS_PER_VECTOR><<<NUM_BLOCKS, 256>>>(d_vecY_csr, d_val, d_indices, d_ptr, d_vecX_csr, vecY->n);
        }
        else if (mean_col_num > 16)
        {
            const int THREADS_PER_VECTOR = 32;
            const unsigned int VECTORS_PER_BLOCK = 8;
            const unsigned int NUM_BLOCKS = static_cast<unsigned int>((vecY->n + (VECTORS_PER_BLOCK - 1)) / VECTORS_PER_BLOCK);
            spmv_kernel_cuda<float, VECTORS_PER_BLOCK, THREADS_PER_VECTOR><<<NUM_BLOCKS, 256>>>(d_vecY_csr, d_val, d_indices, d_ptr, d_vecX_csr, vecY->n);
        }
        // else if (mean_col_num > 32 && mean_col_num <= 64)
        // {
        //     const int THREADS_PER_VECTOR = 64;
        //     const unsigned int VECTORS_PER_BLOCK = 4;
        //     const unsigned int NUM_BLOCKS = static_cast<unsigned int>((vecY->n + (VECTORS_PER_BLOCK - 1)) / VECTORS_PER_BLOCK);
        //     spmv_kernel_cuda_L<float, VECTORS_PER_BLOCK, THREADS_PER_VECTOR><<<NUM_BLOCKS, 256>>>(d_vecY_csr, d_val, d_indices, d_ptr, d_vecX_csr, vecY->n);
        // }
        // else if (mean_col_num > 64)
        // {
        //     const int THREADS_PER_VECTOR = 256;
        //     const unsigned int VECTORS_PER_BLOCK = 1;
        //     const unsigned int NUM_BLOCKS = static_cast<unsigned int>((vecY->n + (VECTORS_PER_BLOCK - 1)) / VECTORS_PER_BLOCK);
        //     spmv_kernel_cuda_L<float, VECTORS_PER_BLOCK, THREADS_PER_VECTOR><<<NUM_BLOCKS, 256>>>(d_vecY_csr, d_val, d_indices, d_ptr, d_vecX_csr, vecY->n);
        // }
    }

    cuda_time_test_end();
    double runtime = (elapsedTime) / test_iter;
    double gflops = (2.0 * matA_csr->nnz) / ((runtime / 1000) * 1e9);
    hipDeviceSynchronize();
    printf("\n SpMV CUDA kernel runtime = %g ms\n", runtime);
    printf("\n SpMV Performance  = %lf GFLOPS\n", gflops);

    hipMemcpy(((float *)vecY_csr->values), d_vecY_csr, sizeof(float) * vecY->n, hipMemcpyDeviceToHost);

    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        printf("Error:%s\n", hipGetErrorString(cudaStatus));
        exit(EXIT_FAILURE);
    }
    hipFree(d_vecY_csr);
    hipFree(d_vecX_csr);
    hipFree(d_val);
    hipFree(d_indices);
    hipFree(d_ptr);

    return perfSpB_success;
}

perfSpB_info perfSpB_SpMV_FP64(perfSpB_operation_t op,
                               const void *alpha,
                               const perfSpB_matrix matA,
                               const perfSpB_vector vecX,
                               const void *beta,
                               perfSpB_vector vecY)
{
    UNUSED(op);
    UNUSED(alpha);
    UNUSED(beta);

    perfSpB_vector_Dense *vecY_csr = (perfSpB_vector_Dense *)(vecY);
    perfSpB_vector_Dense *vecX_csr = (perfSpB_vector_Dense *)(vecX);
    perfSpB_matrix_CSC_or_CSR *matA_csr = (perfSpB_matrix_CSC_or_CSR *)(matA);
    double *d_vecY_csr, *d_vecX_csr, *d_val;
    perfSpB_index *d_indices, *d_ptr;
    for (int i = 0; i < matA->row; i++)
    {
        if (*((double *)vecY_csr->values) != 0)
        {
            printf("\n %f \n", *((double *)vecY_csr->values));
        }
    }

    hipMalloc(&d_vecY_csr, sizeof(double) * vecY->n);
    hipMalloc(&d_vecX_csr, sizeof(double) * vecX->n);
    hipMalloc(&d_val, sizeof(double) * matA_csr->nnz);
    hipMalloc(&d_indices, sizeof(perfSpB_index) * matA_csr->nnz);
    hipMalloc(&d_ptr, sizeof(perfSpB_index) * (vecY->n + 1));

    hipMemcpy(d_val, matA_csr->val, sizeof(double) * matA_csr->nnz, hipMemcpyHostToDevice);
    hipMemcpy(d_indices, matA_csr->indices, sizeof(perfSpB_index) * matA_csr->nnz, hipMemcpyHostToDevice);
    hipMemcpy(d_ptr, matA_csr->ptr, sizeof(perfSpB_index) * (vecY->n + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_vecX_csr, ((double *)vecX_csr->values), sizeof(double) * vecX->n, hipMemcpyHostToDevice);
    
    double mean_col_num = ((double)matA_csr->nnz + (double)(vecY->n - 1)) / (double)vecY->n;
    double accum = 0.0, max = 0.0;
    for (int row_id = 1; row_id < matA->row; row_id++)
    {
        double row_nnz_tmp = (double)matA_csr->ptr[row_id + 1] - (double)matA_csr->ptr[row_id];
        max = (max < row_nnz_tmp) ? row_nnz_tmp : max;
        accum += abs(row_nnz_tmp - mean_col_num);
    }
    cuda_time_test_start();
    for (int i = 0; i < test_iter; i++)
    {
        if (mean_col_num <= 2)
        {
            const int THREADS_PER_VECTOR = 2;
            const unsigned int VECTORS_PER_BLOCK = 128;
            const unsigned int NUM_BLOCKS = static_cast<unsigned int>((vecY->n + (VECTORS_PER_BLOCK - 1)) / VECTORS_PER_BLOCK);
            spmv_kernel_cuda<double, VECTORS_PER_BLOCK, THREADS_PER_VECTOR><<<NUM_BLOCKS, 256>>>(d_vecY_csr, d_val, d_indices, d_ptr, d_vecX_csr, vecY->n);
        }
        else if (mean_col_num > 2 && mean_col_num <= 4)
        {
            const int THREADS_PER_VECTOR = 4;
            const unsigned int VECTORS_PER_BLOCK = 64;
            const unsigned int NUM_BLOCKS = static_cast<unsigned int>((vecY->n + (VECTORS_PER_BLOCK - 1)) / VECTORS_PER_BLOCK);
            spmv_kernel_cuda<double, VECTORS_PER_BLOCK, THREADS_PER_VECTOR><<<NUM_BLOCKS, 256>>>(d_vecY_csr, d_val, d_indices, d_ptr, d_vecX_csr, vecY->n);
        }
        else if (mean_col_num > 4 && mean_col_num <= 8)
        {
            const int THREADS_PER_VECTOR = 8;
            const unsigned int VECTORS_PER_BLOCK = 32;
            const unsigned int NUM_BLOCKS = static_cast<unsigned int>((vecY->n + (VECTORS_PER_BLOCK - 1)) / VECTORS_PER_BLOCK);
            spmv_kernel_cuda<double, VECTORS_PER_BLOCK, THREADS_PER_VECTOR><<<NUM_BLOCKS, 256>>>(d_vecY_csr, d_val, d_indices, d_ptr, d_vecX_csr, vecY->n);
        }
        else if (mean_col_num > 8 && mean_col_num <= 16)
        {
            const int THREADS_PER_VECTOR = 16;
            const unsigned int VECTORS_PER_BLOCK = 16;
            const unsigned int NUM_BLOCKS = static_cast<unsigned int>((vecY->n + (VECTORS_PER_BLOCK - 1)) / VECTORS_PER_BLOCK);
            spmv_kernel_cuda<double, VECTORS_PER_BLOCK, THREADS_PER_VECTOR><<<NUM_BLOCKS, 256>>>(d_vecY_csr, d_val, d_indices, d_ptr, d_vecX_csr, vecY->n);
        }
        else if (mean_col_num > 16)
        {
            const int THREADS_PER_VECTOR = 32;
            const unsigned int VECTORS_PER_BLOCK = 8;
            const unsigned int NUM_BLOCKS = static_cast<unsigned int>((vecY->n + (VECTORS_PER_BLOCK - 1)) / VECTORS_PER_BLOCK);
            spmv_kernel_cuda<double, VECTORS_PER_BLOCK, THREADS_PER_VECTOR><<<NUM_BLOCKS, 256>>>(d_vecY_csr, d_val, d_indices, d_ptr, d_vecX_csr, vecY->n);
        }
    }

    cuda_time_test_end();
    double runtime = (elapsedTime) / test_iter;
    double gflops = (2.0 * matA_csr->nnz) / ((runtime / 1000) * 1e9);
    hipDeviceSynchronize();
    printf("\n SpMV CUDA kernel runtime = %g ms\n", runtime);
    printf("\n SpMV Performance  = %lf GFLOPS\n", gflops);

    hipMemcpy(((double *)vecY_csr->values), d_vecY_csr, sizeof(double) * vecY->n, hipMemcpyDeviceToHost);

    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        printf("Error:%s\n", hipGetErrorString(cudaStatus));
        exit(EXIT_FAILURE);
    }
    hipFree(d_vecY_csr);
    hipFree(d_vecX_csr);
    hipFree(d_val);
    hipFree(d_indices);
    hipFree(d_ptr);

    return perfSpB_success;
}