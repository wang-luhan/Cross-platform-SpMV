#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <spmv_cuda.h>
#include <math.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>

#include "cuda_timer.h"

#define test_iter 2000

#define NUM_ROW_BLOCKS 320
#define L_THREADS 256
// #define G_THREADS 81920

// typedef struct T_INFO
// {
//     perfSpB_coord *thread_coord_start;
//     perfSpB_coord *thread_coord_end;
// } T_INFO;

__global__ void spmv_product_kernel(const perfSpB_index *__restrict__ A_col_index,
    const float *elem,
    const float *__restrict__ x,
    perfSpB_index num_nz,
    float *middle)
{
    unsigned int elem_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (elem_idx < num_nz)
    {
        middle[elem_idx] = elem[elem_idx] * x[A_col_index[elem_idx]];
    }
}


template <typename T>
__global__ void spmv_div_pre(perfSpB_coord *thread_coord_start_array, perfSpB_coord *thread_coord_end_array, perfSpB_index *d_ptr, perfSpB_index d_nnz, perfSpB_index nrows, perfSpB_index *row_block_ptr)
{
    perfSpB_index gid = blockIdx.x * blockDim.x + threadIdx.x;
    perfSpB_index tid = threadIdx.x;

    perfSpB_index block_first_row_id = row_block_ptr[blockIdx.x];
    perfSpB_index block_last_row_id = row_block_ptr[blockIdx.x + 1];
    perfSpB_index block_first_nnz_id = d_ptr[block_first_row_id];
    perfSpB_index block_last_nnz_id = d_ptr[block_last_row_id];

    perfSpB_index block_nnzs = block_last_nnz_id - block_first_nnz_id;
    perfSpB_index block_rows = block_last_row_id - block_first_row_id;

    perfSpB_index thread_nnzs = (block_nnzs + blockDim.x - 1) / blockDim.x;
    perfSpB_index thread_first_nnz_id = min(block_first_nnz_id + tid * thread_nnzs, block_last_nnz_id);
    perfSpB_index thread_last_nnz_id = min(thread_first_nnz_id + thread_nnzs, block_last_nnz_id);
    perfSpB_index thread_first_row_id, thread_last_row_id;
    // if(blockIdx.x == 320)
    // {
    //     printf("\n thread_nnzs = %ld \n", thread_nnzs);
    // }
    for(int i = block_first_row_id; i < block_first_row_id + block_rows; i ++)
    {
        if(thread_first_nnz_id >= d_ptr[i] && thread_first_nnz_id < d_ptr[i + 1])
        {
            thread_first_row_id = i;
            break;
        }
    }

    for(int i = block_first_row_id; i < block_first_row_id + block_rows; i ++)
    {
        if(thread_last_nnz_id >= d_ptr[i] && thread_last_nnz_id < d_ptr[i + 1])
        {
            thread_last_row_id = i;
            break;
        }
    }
    thread_coord_start_array[gid].x = thread_first_row_id;
    thread_coord_start_array[gid].y = thread_first_nnz_id;
    thread_coord_end_array[gid].x = thread_last_row_id;
    thread_coord_end_array[gid].y = thread_last_nnz_id;
}

template <typename T>
__global__ void spmv_merge_kernel_right(perfSpB_coord *thread_coord_start_array,
                                  perfSpB_coord *thread_coord_end_array,
                                  T *d_val,
                                  perfSpB_index *d_ptr,
                                  perfSpB_index *d_cols,
                                  perfSpB_index d_nnz,
                                  T *d_vector,
                                  perfSpB_index nrows,
                                  T *d_out)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    // int tid = threadIdx.x;

    perfSpB_index *row_end_offsets = d_ptr + 1; // Merge list A
    // __shared__ T value_carry_out[L_THREADS];
    // __shared__ int row_carry_out[L_THREADS];
    perfSpB_coord thread_coord_start = thread_coord_start_array[gid];
    perfSpB_coord thread_coord_end = thread_coord_end_array[gid];
    
    for (; thread_coord_start.x < thread_coord_end.x; ++thread_coord_start.x)
    {
        float row_total = 0.0;
        for (; thread_coord_start.y < (int)row_end_offsets[thread_coord_start.x]; ++thread_coord_start.y)
        {
            row_total += d_val[thread_coord_start.y] * d_vector[d_cols[thread_coord_start.y]];
        } // End of this line
        d_out[thread_coord_start.x] = row_total;
    }
    //  finish one row, calculate the partial sum of the next row
    //  Consume partial portion of thread's last row (accumulate any nonzeros for a partial row shared with the next thread)
    float row_partial = 0.0;
    for (; thread_coord_start.y < thread_coord_end.y; ++thread_coord_start.y)
    {
        row_partial += d_val[thread_coord_start.y] * d_vector[d_cols[thread_coord_start.y]];
    }
    __syncthreads();
    /*
    // save the thread's running total and row-id for subsequent fix-up
    row_carry_out[tid] = thread_coord_end.x; // The value of this row is incomplete
    value_carry_out[tid] = row_partial;    // This row would be a little bit more complete with this. Each thread might have one of these
    __syncthreads();
    */
    // atomicAdd(d_out + row_carry_out[tid], value_carry_out[tid]);
    atomicAdd(d_out + thread_coord_end.x, row_partial);
}


template <typename T>
__global__ void spmv_merge_kernel_right_(perfSpB_coord *thread_coord_start_array,
                                  perfSpB_coord *thread_coord_end_array,
                                  T *d_val,
                                  perfSpB_index *d_ptr,
                                  perfSpB_index *d_cols,
                                  perfSpB_index d_nnz,
                                  T *d_vector,
                                  perfSpB_index nrows,
                                  T *d_out)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    // int tid = threadIdx.x;

    perfSpB_index *row_end_offsets = d_ptr + 1; // Merge list A

    perfSpB_coord thread_coord_start = thread_coord_start_array[gid];
    perfSpB_coord thread_coord_end = thread_coord_start_array[gid+1];
    
    for (; thread_coord_start.x < thread_coord_end.x; ++thread_coord_start.x)
    {
        float row_total = 0.0;
        for (; thread_coord_start.y < (int)row_end_offsets[thread_coord_start.x]; ++thread_coord_start.y)
        {
            row_total += d_val[thread_coord_start.y] * d_vector[d_cols[thread_coord_start.y]];
        } // End of this line
        d_out[thread_coord_start.x] = row_total;
    }
    //  finish one row, calculate the partial sum of the next row
    //  Consume partial portion of thread's last row (accumulate any nonzeros for a partial row shared with the next thread)
    float row_partial = 0.0;
    for (; thread_coord_start.y < thread_coord_end.y; ++thread_coord_start.y)
    {
        row_partial += d_val[thread_coord_start.y] * d_vector[d_cols[thread_coord_start.y]];
    }
    __syncthreads();

    atomicAdd(d_out + thread_coord_end.x, row_partial);
}

//TODO: eda实现
template <typename T>
__global__ void spmv_merge_kernel(perfSpB_coord *thread_coord_start_array,
                                  perfSpB_coord *thread_coord_end_array,
                                  perfSpB_index *d_ptr,
                                  T* d_middle,
                                  perfSpB_index d_nnz,
                                  perfSpB_index nrows,
                                  T *d_out)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    // int tid = threadIdx.x;

    perfSpB_index *row_end_offsets = d_ptr + 1; // Merge list A

    perfSpB_coord thread_coord_start = thread_coord_start_array[gid];
    perfSpB_coord thread_coord_end = thread_coord_start_array[gid+1];
    
    for (; thread_coord_start.x < thread_coord_end.x; ++thread_coord_start.x)
    {
        float row_total = 0.0;
        for (; thread_coord_start.y < (int)row_end_offsets[thread_coord_start.x]; ++thread_coord_start.y)
        {
            row_total += d_middle[thread_coord_start.y];
        } // End of this line
        d_out[thread_coord_start.x] = row_total;
    }
    //  finish one row, calculate the partial sum of the next row
    //  Consume partial portion of thread's last row (accumulate any nonzeros for a partial row shared with the next thread)
    float row_partial = 0.0;
    for (; thread_coord_start.y < thread_coord_end.y; ++thread_coord_start.y)
    {
        row_partial += d_middle[thread_coord_start.y];
    }
    __syncthreads();

    atomicAdd(d_out + thread_coord_end.x, row_partial);
}

perfSpB_info perfSpB_SpMV_MERGE_FP32(perfSpB_operation_t op,
                                     const void *alpha,
                                     const perfSpB_matrix matA,
                                     const perfSpB_vector vecX,
                                     const void *beta,
                                     perfSpB_vector vecY)
{
    UNUSED(op);
    UNUSED(alpha);
    UNUSED(beta);
    perfSpB_vector_Dense *vecY_csr = (perfSpB_vector_Dense *)(vecY);
    perfSpB_vector_Dense *vecX_csr = (perfSpB_vector_Dense *)(vecX);
    perfSpB_matrix_CSC_or_CSR *matA_csr = (perfSpB_matrix_CSC_or_CSR *)(matA);
    perfSpB_index nrows = matA->row;
    // perfSpB_index ncols = matA->col;
    perfSpB_index nvals = matA_csr->nnz;

    perfSpB_index NNZ_PER_ROW_BLOCK = ceil(nvals / (NUM_ROW_BLOCKS));
    perfSpB_index *row_block_ptr = (perfSpB_index *)malloc((640 + 2) * sizeof(perfSpB_index));
    perfSpB_index *row_ptr_local = (perfSpB_index *)malloc((vecY->n + 1) * sizeof(perfSpB_index));

    row_block_ptr[0] = 0;
    int b = 0, r = 0, acc = 0;
    int num_row_block_real = NUM_ROW_BLOCKS;

    for (b = 1; b <= 640; b++)
    {
        acc = matA_csr->ptr[r + 1] - matA_csr->ptr[r];// 第r行首先包含在第(b-1)个行块中
        while (r < nrows)
        {
            r++;
            acc += matA_csr->ptr[r + 1] - matA_csr->ptr[r];// 看r+1行是否包含在这个块，若不包含，r+1赋值给row_block_ptr[b]
            if (acc >= NNZ_PER_ROW_BLOCK)
            {
                break;
            }
        }
        row_block_ptr[b] = r;
        if (r >= nrows)
        {
            num_row_block_real = b;
            // printf("\n 666666  num_row_block_real = %d\n", num_row_block_real);
            break;
        }
    }
    // printf("\n r = %d  nrows = %ld \n", r, nrows);

    for (int b = 0; b < num_row_block_real; b++) // 遍历每一个用到的块
    {
        perfSpB_index block_first_nnz_id = matA_csr->ptr[row_block_ptr[b]]; // 每块第一个非零元id
        for (int i = row_block_ptr[b]; i < row_block_ptr[b + 1]; i++)// 遍历b块的每一行
        {
            row_ptr_local[i] = matA_csr->ptr[i] - block_first_nnz_id;// 0 开始的块内行偏移
        }
    }
  

    // for (b = 0; b < num_row_block_real; b++)
    // {
    //     printf("\n [block%d], row%ld-row%ld  nnz= %ld\n", b, row_block_ptr[b], row_block_ptr[b + 1], matA_csr->ptr[row_block_ptr[b + 1]] - matA_csr->ptr[row_block_ptr[b]]);
    // }


    float *d_vecY_csr, *d_vecX_csr, *d_val, *d_middle;
    perfSpB_index *d_indices, *d_ptr, *d_row_block_ptr, *d_ptr_local;
    for (int i = 0; i < matA->row; i++)
    {
        if (*((float *)vecY_csr->values) != 0)
        {
            printf("\n %f \n", *((float *)vecY_csr->values));
        }
    }

    hipMalloc(&d_vecY_csr, sizeof(float) * vecY->n);
    hipMalloc(&d_vecX_csr, sizeof(float) * vecX->n);
    hipMalloc(&d_val, sizeof(float) * matA_csr->nnz);
    hipMalloc(&d_indices, sizeof(perfSpB_index) * matA_csr->nnz);
    hipMalloc(&d_ptr, sizeof(perfSpB_index) * (vecY->n + 1));
    hipMalloc(&d_ptr_local, sizeof(perfSpB_index) * (vecY->n + 1));
    hipMalloc(&d_row_block_ptr, (640 + 2) * sizeof(perfSpB_index));
    hipMalloc(&d_middle, sizeof(float) * matA_csr->nnz);

    hipMemcpy(d_val, matA_csr->val, sizeof(float) * matA_csr->nnz, hipMemcpyHostToDevice);
    hipMemcpy(d_indices, matA_csr->indices, sizeof(perfSpB_index) * matA_csr->nnz, hipMemcpyHostToDevice);
    hipMemcpy(d_ptr, matA_csr->ptr, sizeof(perfSpB_index) * (vecY->n + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_ptr_local, row_ptr_local, sizeof(perfSpB_index) * (vecY->n + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_vecX_csr, ((float *)vecX_csr->values), sizeof(float) * vecX->n, hipMemcpyHostToDevice);
    hipMemcpy(d_row_block_ptr, row_block_ptr, (640 + 2) * sizeof(perfSpB_index), hipMemcpyHostToDevice);
    hipMemset(d_middle, 0.0, sizeof(float) * (matA_csr->nnz));

    perfSpB_coord *thread_coord_start_array;
    perfSpB_coord *thread_coord_end_array;
    perfSpB_index G_THREADS = num_row_block_real * L_THREADS;

    perfSpB_coord *thread_break = (perfSpB_coord *)malloc((G_THREADS + 1) * sizeof(perfSpB_coord));
    for(int i = 0; i < G_THREADS + 1; i++)
    {
        thread_break[i].x = nrows;
        thread_break[i].y = nvals;
    }



    hipMalloc((void **)&thread_coord_start_array, sizeof(perfSpB_coord) * (G_THREADS + 1));
    hipMalloc((void **)&thread_coord_end_array, sizeof(perfSpB_coord) * G_THREADS);
    // hipMemset(thread_coord_start_array, 0, sizeof(perfSpB_coord) * G_THREADS);
    hipMemset(thread_coord_end_array, 0, sizeof(perfSpB_coord) * G_THREADS);
    hipMemcpy(thread_coord_start_array, thread_break, sizeof(perfSpB_coord) * (G_THREADS + 1), hipMemcpyHostToDevice);
    

    
    spmv_div_pre<float><<<num_row_block_real, L_THREADS>>>(thread_coord_start_array, thread_coord_end_array, d_ptr, nvals, nrows, d_row_block_ptr);

    /////////////////////////////////////pre test/////////////////////////////////////
    perfSpB_coord *t_coo_start_array = (perfSpB_coord *)malloc(sizeof(perfSpB_coord) * G_THREADS);
    perfSpB_coord *t_coo_end_array = (perfSpB_coord *)malloc(sizeof(perfSpB_coord) * G_THREADS);
    hipMemcpy(t_coo_start_array, thread_coord_start_array, sizeof(perfSpB_coord) * G_THREADS, hipMemcpyDeviceToHost);
    hipMemcpy(t_coo_end_array, thread_coord_end_array, sizeof(perfSpB_coord) * G_THREADS, hipMemcpyDeviceToHost);
    // for (int i = (num_row_block_real - 1) * L_THREADS; i < num_row_block_real * L_THREADS; i++)
    // {
    //     printf("\n [%d][%d] x_start = %d, x_end = %d, y_start = %d, y_end = %d \n",(int)(i/L_THREADS),i, t_coo_start_array[i].x, t_coo_end_array[i].x, t_coo_start_array[i].y, t_coo_end_array[i].y) ;
    // }
    // printf("\n 400725 %ld ~ %ld \n",matA_csr->ptr[400725], matA_csr->ptr[400726]);
    // printf("\n 400726 %ld ~ %ld \n",matA_csr->ptr[400726], matA_csr->ptr[400727]);
    // printf("\n 2 %ld ~ %ld \n",matA_csr->ptr[2], matA_csr->ptr[3]);

    //TODO: 写一个测试t_coo_start_array与t_coo_end_array的脚本
   

    // for (int i = 0; i < num_row_block_real * L_THREADS - 1; i++)
    // {
    //     if((t_coo_end_array[i].x != t_coo_start_array[i+1].x) || (t_coo_end_array[i].y != t_coo_start_array[i+1].y))
    //     {
    //         printf("\n coo div error!!!!, [%d][%d] x_start = %d, x_end = %d, y_start = %d, y_end = %d \n", (int)(i/L_THREADS),i, t_coo_start_array[i].x, t_coo_end_array[i].x, t_coo_start_array[i].y, t_coo_end_array[i].y);
    //         printf("\n coo div error!!!!, [%d][%d] x_start = %d, x_end = %d, y_start = %d, y_end = %d \n", (int)((i+1)/L_THREADS),i+1, t_coo_start_array[i+1].x, t_coo_end_array[i+1].x, t_coo_start_array[i+1].y, t_coo_end_array[i+1].y);
    //     }
    //     // if((t_coo_end_array[i].x == t_coo_start_array[i].x) )
    //     // {
    //     //     printf("\n !!!!, [%d]\n", t_coo_start_array[i].x);
    //     //     // if(t_coo_start_array[i].x > 15000)
    //     //     // {
    //     //     //     break;
    //     //     // }
    //     // }
    // }
     /////////////////////////////////////pre test/////////////////////////////////////
    const unsigned int NUM_BLOCKS = static_cast<unsigned int>((matA_csr->nnz + (1024 - 1)) / 1024);

    cuda_time_test_start();
    for (int i = 0; i < test_iter; i++)
    {
        spmv_product_kernel<<<NUM_BLOCKS, 1024>>>(d_indices, d_val, d_vecX_csr, matA_csr->nnz, d_middle);
        spmv_merge_kernel<float><<<num_row_block_real, L_THREADS>>>(thread_coord_start_array, thread_coord_end_array, d_ptr, d_middle, nvals, vecY->n, d_vecY_csr);
    }
    cuda_time_test_end();
    double runtime = (elapsedTime) / test_iter;
    double gflops = (2.0 * matA_csr->nnz) / ((runtime / 1000) * 1e9);

    hipDeviceSynchronize();
    printf("\n SpMV CUDA kernel runtime = %g ms\n", runtime);
    // printf("\n SpMV CUDA kernel2 runtime = %g ms\n", runtime2);
    printf("\n SpMV Performance  = %lf GFLOPS\n", gflops);

    hipMemcpy(((float *)vecY_csr->values), d_vecY_csr, sizeof(float) * vecY->n, hipMemcpyDeviceToHost);

    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        printf("Error:%s\n", hipGetErrorString(cudaStatus));
        exit(EXIT_FAILURE);
    }
    hipFree(thread_coord_start_array);
    hipFree(thread_coord_end_array);
    hipFree(d_vecY_csr);
    hipFree(d_vecX_csr);
    hipFree(d_val);
    hipFree(d_indices);
    hipFree(d_ptr);

    return perfSpB_success;
}
