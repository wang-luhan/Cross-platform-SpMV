#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <spmv_cuda.h>
#include <math.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>

#include "cuda_timer.h"

#define test_iter 2000

#define threadsPerBlock 256
#define sizeSharedMemory 1024
#define BlockDim 1024

template <typename T>
__global__ void spmv_pcsr_kernel1(T *d_val, T *d_vector, perfSpB_index *d_cols, perfSpB_index d_nnz, T *d_middle)
{
    perfSpB_index tid = blockIdx.x * blockDim.x + threadIdx.x;
    perfSpB_index icr = blockDim.x * gridDim.x;
    while (tid < d_nnz)
    {
        d_middle[tid] = d_val[tid] * d_vector[d_cols[tid]];
        tid += icr;
    }
}

template <typename T>
__global__ void spmv_pcsr_kernel2(T *d_middle, perfSpB_index *d_ptr, perfSpB_index nrows, T *d_out)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    __shared__ volatile perfSpB_index ptr_s[threadsPerBlock + 1];
    __shared__ volatile T middle_s[sizeSharedMemory];

    // Load ptr into the shared memory ptr_s
    ptr_s[tid] = d_ptr[gid];

    // Assign thread 0 of every block to store the pointer for the last row handled by the block into the last shared memory location
    if (tid == 0)
    {
        if (gid + threadsPerBlock > nrows)
        {
            ptr_s[threadsPerBlock] = d_ptr[nrows];
        }
        else
        {
            ptr_s[threadsPerBlock] = d_ptr[gid + threadsPerBlock];
        }
    }
    __syncthreads();

    perfSpB_index temp = (ptr_s[threadsPerBlock] - ptr_s[0]) / threadsPerBlock + 1;
    perfSpB_index nlen = min(perfSpB_index(temp * threadsPerBlock), perfSpB_index(sizeSharedMemory));
    T sum = 0;
    perfSpB_index maxlen = ptr_s[threadsPerBlock];
    //使用循环分多次加载 d_middle 到共享内存 middle_s[],每次加载量为 nlen
    for (perfSpB_index i = ptr_s[0]; i < maxlen; i += nlen)
    {
        perfSpB_index index = i + tid;
        __syncthreads();
        // Load d_middle into the shared memory middle_s, 填满一个1024需要几次迭代
        for (perfSpB_index j = 0; j < nlen / threadsPerBlock; j++)
        {
            if (index < maxlen)
            {
                middle_s[tid + j * threadsPerBlock] = d_middle[index];
                index += threadsPerBlock;
            }
        }
        __syncthreads();
        //填满了一个1024，计算这个1024
        // Sum up the elements for a row
        if (ptr_s[tid + 1] > i && ptr_s[tid] <= i + nlen - 1)
        {
            perfSpB_index row_s = max(perfSpB_index(ptr_s[tid] - i), perfSpB_index(0));
            perfSpB_index row_e = min(perfSpB_index(ptr_s[tid + 1] - i), perfSpB_index(nlen));
            for (perfSpB_index j = row_s; j < row_e; j++)
            {
                sum += middle_s[j];
            }
        }
    }
    // Write result
    d_out[gid] = sum;
}

/*
template <typename T>
__global__ void spmv_pcsr_kernel2(T *d_middle, perfSpB_index *d_ptr, perfSpB_index nrows, T *d_out)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    int b_0_id = blockIdx.x * blockDim.x;

    __shared__ volatile T middle_s[sizeSharedMemory];

    int b_last_id = (b_0_id + threadsPerBlock) > nrows ? nrows : (b_0_id + threadsPerBlock);

    perfSpB_index temp = (d_ptr[b_last_id] - d_ptr[b_0_id]) / threadsPerBlock + 1;
    perfSpB_index nlen = min(perfSpB_index(temp * threadsPerBlock), perfSpB_index(sizeSharedMemory));
    T sum = 0;
    perfSpB_index maxlen = d_ptr[b_last_id];
    // 使用循环分多次加载 d_middle 到共享内存 middle_s[],每次加载量为 nlen
    for (perfSpB_index i = d_ptr[b_0_id]; i < maxlen; i += nlen)
    {
        perfSpB_index index = i + tid;
        __syncthreads();
        // Load d_middle into the shared memory middle_s, 填满一个1024需要几次迭代
        for (perfSpB_index j = 0; j < nlen / threadsPerBlock; j++)
        {
            if (index < maxlen)
            {
                middle_s[tid + j * threadsPerBlock] = d_middle[index];
                index += threadsPerBlock;
            }
        }
        __syncthreads();
        // 填满了一个1024，计算这个1024
        //  Sum up the elements for a row
        if (d_ptr[gid + 1] > i && d_ptr[gid] <= i + nlen - 1)
        {
            perfSpB_index row_s = max(perfSpB_index(d_ptr[gid] - i), perfSpB_index(0));
            perfSpB_index row_e = min(perfSpB_index(d_ptr[gid + 1] - i), perfSpB_index(nlen));
            for (perfSpB_index j = row_s; j < row_e; j++)
            {
                sum += middle_s[j];
            }
        }
    }
    // Write result
    d_out[gid] = sum;
}
*/
perfSpB_info perfSpB_SpMV_PCSR_FP32(perfSpB_operation_t op,
                                    const void *alpha,
                                    const perfSpB_matrix matA,
                                    const perfSpB_vector vecX,
                                    const void *beta,
                                    perfSpB_vector vecY)
{
    UNUSED(op);
    UNUSED(alpha);
    UNUSED(beta);
    perfSpB_vector_Dense *vecY_csr = (perfSpB_vector_Dense *)(vecY);
    perfSpB_vector_Dense *vecX_csr = (perfSpB_vector_Dense *)(vecX);
    perfSpB_matrix_CSC_or_CSR *matA_csr = (perfSpB_matrix_CSC_or_CSR *)(matA);
    float *d_vecY_csr, *d_vecX_csr, *d_val, *d_v;
    perfSpB_index *d_indices, *d_ptr;
    for (int i = 0; i < matA->row; i++)
    {
        if (*((float *)vecY_csr->values) != 0)
        {
            printf("\n %f \n", *((float *)vecY_csr->values));
        }
    }

    hipMalloc(&d_vecY_csr, sizeof(float) * vecY->n);
    hipMalloc(&d_vecX_csr, sizeof(float) * vecX->n);
    hipMalloc(&d_val, sizeof(float) * matA_csr->nnz);
    hipMalloc(&d_v, sizeof(float) * matA_csr->nnz);
    hipMalloc(&d_indices, sizeof(perfSpB_index) * matA_csr->nnz);
    hipMalloc(&d_ptr, sizeof(perfSpB_index) * (vecY->n + 1));

    hipMemcpy(d_val, matA_csr->val, sizeof(float) * matA_csr->nnz, hipMemcpyHostToDevice);
    hipMemcpy(d_indices, matA_csr->indices, sizeof(perfSpB_index) * matA_csr->nnz, hipMemcpyHostToDevice);
    hipMemcpy(d_ptr, matA_csr->ptr, sizeof(perfSpB_index) * (vecY->n + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_vecX_csr, ((float *)vecX_csr->values), sizeof(float) * vecX->n, hipMemcpyHostToDevice);

    cuda_time_test_start();
    for (int i = 0; i < test_iter; i++)
    {
        spmv_pcsr_kernel1<float><<<ceil(matA_csr->nnz / (float)BlockDim), BlockDim>>>(d_val, d_vecX_csr, d_indices, matA_csr->nnz, d_v);
        spmv_pcsr_kernel2<float><<<ceil(vecY->n / (float)threadsPerBlock), threadsPerBlock>>>(d_v, d_ptr, vecY->n, d_vecY_csr);
    }
    cuda_time_test_end();
    double runtime1 = (elapsedTime) / test_iter;
    double gflops = (2.0 * matA_csr->nnz) / ((runtime1 / 1000) * 1e9);

    // cuda_time_test_start();
    // for (int i = 0; i < test_iter; i++)
    // {
    //     // spmv_pcsr_kernel1<float><<<ceil(matA_csr->nnz / (float)BlockDim), BlockDim>>>(d_val, d_vecX_csr, d_indices, matA_csr->nnz, d_v);
    // 	spmv_pcsr_kernel2<float><<<ceil(vecY->n / (float)threadsPerBlock), threadsPerBlock>>>(d_v, d_ptr, vecY->n, d_vecY_csr);
    // }
    // cuda_time_test_end();
    // double runtime2 = (elapsedTime) / test_iter;

    hipDeviceSynchronize();
    printf("\n SpMV CUDA kernel1 runtime = %g ms\n", runtime1);
    // printf("\n SpMV CUDA kernel2 runtime = %g ms\n", runtime2);
    printf("\n SpMV Performance  = %lf GFLOPS\n", gflops);

    hipMemcpy(((float *)vecY_csr->values), d_vecY_csr, sizeof(float) * vecY->n, hipMemcpyDeviceToHost);

    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        printf("Error:%s\n", hipGetErrorString(cudaStatus));
        exit(EXIT_FAILURE);
    }
    hipFree(d_vecY_csr);
    hipFree(d_vecX_csr);
    hipFree(d_val);
    hipFree(d_indices);
    hipFree(d_ptr);

    return perfSpB_success;
}

perfSpB_info perfSpB_SpMV_PCSR_FP64(perfSpB_operation_t op,
                                    const void *alpha,
                                    const perfSpB_matrix matA,
                                    const perfSpB_vector vecX,
                                    const void *beta,
                                    perfSpB_vector vecY)
{
    UNUSED(op);
    UNUSED(alpha);
    UNUSED(beta);
    perfSpB_vector_Dense *vecY_csr = (perfSpB_vector_Dense *)(vecY);
    perfSpB_vector_Dense *vecX_csr = (perfSpB_vector_Dense *)(vecX);
    perfSpB_matrix_CSC_or_CSR *matA_csr = (perfSpB_matrix_CSC_or_CSR *)(matA);
    double *d_vecY_csr, *d_vecX_csr, *d_val, *d_v;
    perfSpB_index *d_indices, *d_ptr;
    for (int i = 0; i < matA->row; i++)
    {
        if (*((double *)vecY_csr->values) != 0)
        {
            printf("\n %f \n", *((double *)vecY_csr->values));
        }
    }

    hipMalloc(&d_vecY_csr, sizeof(double) * vecY->n);
    hipMalloc(&d_vecX_csr, sizeof(double) * vecX->n);
    hipMalloc(&d_val, sizeof(double) * matA_csr->nnz);
    hipMalloc(&d_v, sizeof(double) * matA_csr->nnz);
    hipMalloc(&d_indices, sizeof(perfSpB_index) * matA_csr->nnz);
    hipMalloc(&d_ptr, sizeof(perfSpB_index) * (vecY->n + 1));

    hipMemcpy(d_val, matA_csr->val, sizeof(double) * matA_csr->nnz, hipMemcpyHostToDevice);
    hipMemcpy(d_indices, matA_csr->indices, sizeof(perfSpB_index) * matA_csr->nnz, hipMemcpyHostToDevice);
    hipMemcpy(d_ptr, matA_csr->ptr, sizeof(perfSpB_index) * (vecY->n + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_vecX_csr, ((double *)vecX_csr->values), sizeof(double) * vecX->n, hipMemcpyHostToDevice);

    cuda_time_test_start();
    for (int i = 0; i < test_iter; i++)
    {
        spmv_pcsr_kernel1<double><<<ceil(matA_csr->nnz / (double)BlockDim), BlockDim>>>(d_val, d_vecX_csr, d_indices, matA_csr->nnz, d_v);
        spmv_pcsr_kernel2<double><<<ceil(vecY->n / (double)threadsPerBlock), threadsPerBlock>>>(d_v, d_ptr, vecY->n, d_vecY_csr);
    }
    cuda_time_test_end();
    double runtime1 = (elapsedTime) / test_iter;
    double gflops = (2.0 * matA_csr->nnz) / ((runtime1 / 1000) * 1e9);

    hipDeviceSynchronize();
    printf("\n SpMV CUDA kernel1 runtime = %g ms\n", runtime1);
    // printf("\n SpMV CUDA kernel2 runtime = %g ms\n", runtime2);
    printf("\n SpMV Performance  = %lf GFLOPS\n", gflops);

    hipMemcpy(((double *)vecY_csr->values), d_vecY_csr, sizeof(double) * vecY->n, hipMemcpyDeviceToHost);

    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        printf("Error:%s\n", hipGetErrorString(cudaStatus));
        exit(EXIT_FAILURE);
    }
    hipFree(d_vecY_csr);
    hipFree(d_vecX_csr);
    hipFree(d_val);
    hipFree(d_indices);
    hipFree(d_ptr);

    return perfSpB_success;
}
