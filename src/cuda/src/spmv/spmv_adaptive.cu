#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <spmv_cuda.h>
#include <math.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>

#include "cuda_timer.h"

#define test_iter 2000

#define warpSize 32

#define BlockDim 1024

#define ITER 3

template <typename T>
__global__ void spmv_csr_adaptive_kernel(T * d_val,T * d_vector,perfSpB_index * d_cols,perfSpB_index * d_ptr,perfSpB_index N, perfSpB_index * d_rowBlocks, T * d_out)
{
    	perfSpB_index startRow = d_rowBlocks[blockIdx.x];
    	perfSpB_index nextStartRow = d_rowBlocks[blockIdx.x + 1];
    	perfSpB_index num_rows = nextStartRow -  startRow;
    	perfSpB_index i = threadIdx.x;
    	__shared__ volatile T LDS[BlockDim];
    	// If the block consists of more than one row then run CSR Stream
    	if (num_rows > 1) {
		perfSpB_index nnz = d_ptr[nextStartRow] - d_ptr[startRow];
        	perfSpB_index first_col = d_ptr[startRow];

		// Each thread writes to shared memory
        	if (i < nnz)
		{
			LDS[i] = d_val[first_col + i] * d_vector[d_cols[first_col + i]];
		}
		__syncthreads();     
		
		// Threads that fall within a range sum up the partial results
		for (perfSpB_index k = startRow + i; k < nextStartRow; k += blockDim.x)
		{
			T temp = 0;
			for (perfSpB_index j= (d_ptr[k] - first_col); j < (d_ptr[k + 1] - first_col); j++){
		    		temp = temp + LDS[j];
			}
			d_out[k] = temp;
    		}
	}
	// If the block consists of only one row then run CSR Vector
    	else {
    		// Thread ID in warp
        	perfSpB_index rowStart = d_ptr[startRow];
        	perfSpB_index rowEnd = d_ptr[nextStartRow];

        	T sum = 0;

		// Use all threads in a warp to accumulate multiplied elements
        	for (perfSpB_index j = rowStart + i; j < rowEnd; j += BlockDim)
        	{
            		perfSpB_index col = d_cols[j];
            		sum += d_val[j] * d_vector[col];
        	}

        	LDS[i] = sum;
		__syncthreads();

        	// Reduce partial sums
		for (perfSpB_index stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
			__syncthreads();
			if (i < stride)
				LDS[i] += LDS[i + stride]; 
		}
        	// Write result
        	if (i == 0)
           		d_out[startRow] = LDS[i];
    	}	
}


// calculates the number of rows of a CSR matrix that can fit perfSpB_indexo BlockDim entries.
perfSpB_index spmv_csr_adaptive_rowblocks(perfSpB_index *ptr,perfSpB_index totalRows,perfSpB_index *rowBlocks)
{
    	rowBlocks[0] = 0; 
    	perfSpB_index sum = 0; 
    	perfSpB_index last_i = 0; 
    	perfSpB_index ctr = 1;
    	for (perfSpB_index i = 1; i < totalRows; i++) {
    	// Count non-zeroes in this row 
        	sum += ptr[i] - ptr[i-1];
        	if (sum == BlockDim){
            		// This row fills up LOCAL_SIZE 
            		last_i = i;
            		rowBlocks[ctr++] = i;
            		sum = 0;
        	}
        	else if (sum > BlockDim){
        		if (i - last_i > 1) {
                	// This extra row will not fit 
                		rowBlocks[ctr++] = i - 1;
                		i--;
            		}
            		else if (i - last_i == 1)
                		// This one row is too large
                		rowBlocks[ctr++] = i;
        	last_i = i;
        	sum = 0;
        	}
    	}
    	rowBlocks[ctr++] = totalRows;
    	return ctr;
}

perfSpB_info perfSpB_SpMV_Adaptive_FP32(perfSpB_operation_t op,
                                        const void *alpha,
                                        const perfSpB_matrix matA,
                                        const perfSpB_vector vecX,
                                        const void *beta,
                                        perfSpB_vector vecY)
{
    UNUSED(op);
    UNUSED(alpha);
    UNUSED(beta);
    perfSpB_vector_Dense *vecY_csr = (perfSpB_vector_Dense *)(vecY);
    perfSpB_vector_Dense *vecX_csr = (perfSpB_vector_Dense *)(vecX);
    perfSpB_matrix_CSC_or_CSR *matA_csr = (perfSpB_matrix_CSC_or_CSR *)(matA);
    float *d_vecY_csr, *d_vecX_csr, *d_val;
    perfSpB_index *d_indices, *d_ptr;
    for (perfSpB_index i = 0; i < matA->row; i++)
    {
        if (*((float *)vecY_csr->values) != 0)
        {
            printf("\n %f \n", *((float *)vecY_csr->values));
        }
    }

    hipMalloc(&d_vecY_csr, sizeof(float) * vecY->n);
    hipMalloc(&d_vecX_csr, sizeof(float) * vecX->n);
    hipMalloc(&d_val, sizeof(float) * matA_csr->nnz);

    hipMalloc(&d_indices, sizeof(perfSpB_index) * matA_csr->nnz);
    hipMalloc(&d_ptr, sizeof(perfSpB_index) * (vecY->n + 1));

    hipMemcpy(d_val, matA_csr->val, sizeof(float) * matA_csr->nnz, hipMemcpyHostToDevice);
    hipMemcpy(d_indices, matA_csr->indices, sizeof(perfSpB_index) * matA_csr->nnz, hipMemcpyHostToDevice);
    hipMemcpy(d_ptr, matA_csr->ptr, sizeof(perfSpB_index) * (vecY->n + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_vecX_csr, ((float *)vecX_csr->values), sizeof(float) * vecX->n, hipMemcpyHostToDevice);

    perfSpB_index *rowBlocks, *d_rowBlocks;
    rowBlocks = (perfSpB_index *)malloc((vecY->n) * sizeof(perfSpB_index));
    perfSpB_index countRowBlocks = spmv_csr_adaptive_rowblocks(matA_csr->ptr, vecY->n, rowBlocks);
    hipMalloc(&d_rowBlocks, countRowBlocks * sizeof(perfSpB_index));
    hipMemcpy(d_rowBlocks, rowBlocks, countRowBlocks * sizeof(perfSpB_index), hipMemcpyHostToDevice);
    cuda_time_test_start();

    for (perfSpB_index i = 0; i < test_iter; i++)
    {
        spmv_csr_adaptive_kernel<float><<<(countRowBlocks - 1), BlockDim>>>(d_val, d_vecX_csr, d_indices, d_ptr, vecY->n, d_rowBlocks, d_vecY_csr);
    }

    cuda_time_test_end();
    double runtime1 = (elapsedTime) / test_iter;
    double gflops = (2.0 * matA_csr->nnz) / ((runtime1 / 1000) * 1e9);

    hipDeviceSynchronize();
    printf("\n ------------------------------------------------------------------------------");
    printf("\n SpMV CUDA kernel runtime = %g ms\n", runtime1);
    // printf("\n SpMV CUDA kernel2 runtime = %g ms\n", runtime2);
    printf("\n SpMV Performance  = %lf GFLOPS\n", gflops);
    printf(" ------------------------------------------------------------------------------ \n");

    hipMemcpy(((float *)vecY_csr->values), d_vecY_csr, sizeof(float) * vecY->n, hipMemcpyDeviceToHost);

    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        printf("Error:%s\n", hipGetErrorString(cudaStatus));
        exit(EXIT_FAILURE);
    }
    free(rowBlocks);
    hipFree(d_rowBlocks);
    hipFree(d_vecY_csr);
    hipFree(d_vecX_csr);
    hipFree(d_val);
    hipFree(d_indices);
    hipFree(d_ptr);

    return perfSpB_success;
}

perfSpB_info perfSpB_SpMV_Adaptive_FP64(perfSpB_operation_t op,
                                        const void *alpha,
                                        const perfSpB_matrix matA,
                                        const perfSpB_vector vecX,
                                        const void *beta,
                                        perfSpB_vector vecY)
{
    UNUSED(op);
    UNUSED(alpha);
    UNUSED(beta);
    perfSpB_vector_Dense *vecY_csr = (perfSpB_vector_Dense *)(vecY);
    perfSpB_vector_Dense *vecX_csr = (perfSpB_vector_Dense *)(vecX);
    perfSpB_matrix_CSC_or_CSR *matA_csr = (perfSpB_matrix_CSC_or_CSR *)(matA);
    double *d_vecY_csr, *d_vecX_csr, *d_val;
    perfSpB_index *d_indices, *d_ptr;
    for (perfSpB_index i = 0; i < matA->row; i++)
    {
        if (*((double *)vecY_csr->values) != 0)
        {
            printf("\n %f \n", *((double *)vecY_csr->values));
        }
    }

    hipMalloc(&d_vecY_csr, sizeof(double) * vecY->n);
    hipMalloc(&d_vecX_csr, sizeof(double) * vecX->n);
    hipMalloc(&d_val, sizeof(double) * matA_csr->nnz);

    hipMalloc(&d_indices, sizeof(perfSpB_index) * matA_csr->nnz);
    hipMalloc(&d_ptr, sizeof(perfSpB_index) * (vecY->n + 1));

    hipMemcpy(d_val, matA_csr->val, sizeof(double) * matA_csr->nnz, hipMemcpyHostToDevice);
    hipMemcpy(d_indices, matA_csr->indices, sizeof(perfSpB_index) * matA_csr->nnz, hipMemcpyHostToDevice);
    hipMemcpy(d_ptr, matA_csr->ptr, sizeof(perfSpB_index) * (vecY->n + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_vecX_csr, ((double *)vecX_csr->values), sizeof(double) * vecX->n, hipMemcpyHostToDevice);

    perfSpB_index *rowBlocks, *d_rowBlocks;
    rowBlocks = (perfSpB_index *)malloc((vecY->n) * sizeof(perfSpB_index));
    perfSpB_index countRowBlocks = spmv_csr_adaptive_rowblocks(matA_csr->ptr, vecY->n, rowBlocks);
    hipMalloc(&d_rowBlocks, countRowBlocks * sizeof(perfSpB_index));
    hipMemcpy(d_rowBlocks, rowBlocks, countRowBlocks * sizeof(perfSpB_index), hipMemcpyHostToDevice);
    cuda_time_test_start();

    for (perfSpB_index i = 0; i < test_iter; i++)
    {
        spmv_csr_adaptive_kernel<double><<<(countRowBlocks - 1), BlockDim>>>(d_val, d_vecX_csr, d_indices, d_ptr, vecY->n, d_rowBlocks, d_vecY_csr);
    }

    cuda_time_test_end();
    double runtime1 = (elapsedTime) / test_iter;
    double gflops = (2.0 * matA_csr->nnz) / ((runtime1 / 1000) * 1e9);

    hipDeviceSynchronize();
    printf("\n ------------------------------------------------------------------------------");
    printf("\n SpMV CUDA kernel runtime = %g ms\n", runtime1);
    // printf("\n SpMV CUDA kernel2 runtime = %g ms\n", runtime2);
    printf("\n SpMV Performance  = %lf GFLOPS\n", gflops);
    printf(" ------------------------------------------------------------------------------ \n");

    hipMemcpy(((double *)vecY_csr->values), d_vecY_csr, sizeof(double) * vecY->n, hipMemcpyDeviceToHost);

    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        printf("Error:%s\n", hipGetErrorString(cudaStatus));
        exit(EXIT_FAILURE);
    }
    free(rowBlocks);
    hipFree(d_rowBlocks);
    hipFree(d_vecY_csr);
    hipFree(d_vecX_csr);
    hipFree(d_val);
    hipFree(d_indices);
    hipFree(d_ptr);

    return perfSpB_success;
}
